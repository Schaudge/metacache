#include "hip/hip_runtime.h"
#include "gpu_hashmap.cuh"
#include "hash_dna.h"
#include "hash_int.h"
#include "sketch_database.h"
#include "gpu_engine.cuh"

namespace mc {

    __global__
    void insert_kernel(target_id tgt, encodedseq_t* encodedSeq, encodedambig_t* encodedAmbig) {

    }


    //-----------------------------------------------------
    template<
        class Key,
        class ValueT,
        class Hash,
        class KeyEqual,
        class BucketSizeT
    >
    void gpu_hashmap<Key,ValueT,Hash,KeyEqual,BucketSizeT>::init()
    {
        hipSetDevice(0);
    }


    //-----------------------------------------------------
    template<
        class Key,
        class ValueT,
        class Hash,
        class KeyEqual,
        class BucketSizeT
    >
    void gpu_hashmap<Key,ValueT,Hash,KeyEqual,BucketSizeT>::insert(
        target_id tgt,
        std::vector<encodedseq_t> encodedSeq,
        std::vector<encodedambig_t> encodedAmbig,
        numk_t k)
    {
        encodedseq_t * d_encodedSeq;
        encodedambig_t * d_encodedAmbig;
        size_t encodedLength = encodedSeq.size();

        hipMalloc(&d_encodedSeq, encodedLength*sizeof(encodedseq_t));
        hipMalloc(&d_encodedAmbig, encodedLength*sizeof(encodedambig_t));

        kmer_type * h_kmers, * d_kmers;
        size_t numKmers = encodedLength*sizeof(encodedambig_t)*CHAR_BIT-k+1;
        uint64_t * d_kmerCounter;

        hipHostMalloc(&h_kmers, numKmers*sizeof(kmer_type));
        hipMalloc(&d_kmers, numKmers*sizeof(kmer_type));
        hipMalloc(&d_kmerCounter, sizeof(uint64_t));

        hipMemcpy(d_encodedSeq, encodedSeq.data(),
                   encodedLength*sizeof(encodedseq_t), hipMemcpyHostToDevice);
        hipMemcpy(d_encodedAmbig, encodedAmbig.data(),
                   encodedLength*sizeof(encodedambig_t), hipMemcpyHostToDevice);

        // insert_kernel<<<1,1>>>(tgt, d_encodedSeq, d_encodedAmbig);

        extract_kmers<<<1024,1024>>>(d_encodedSeq, d_encodedAmbig, encodedLength,
                                     k, d_kmers, d_kmerCounter);

        hipMemcpy(h_kmers, d_kmers, numKmers*sizeof(kmer_type), hipMemcpyDeviceToHost);

        std::cout << "Target ID: " << tgt << '\n';

        //print kmers
        for(size_t i=0; i<numKmers; ++i) {
            std:: cout << h_kmers[i] << ' ';
        }
        std::cout << std::endl;
    }





    //-----------------------------------------------------
    template class gpu_hashmap<
            unsigned int,
            // sketch_database<
            //     std::__cxx11::basic_string<
            //         char,
            //         std::char_traits<char>,
            //         std::allocator<char> >,
            //     single_function_unique_min_hasher<
            //         unsigned int, same_size_hash<unsigned int> >,
            //     same_size_hash<unsigned int>,
            //     unsigned short,
            //     unsigned int,
            //     unsigned char
            //     >::target_location,
            uint64_t,
            same_size_hash<unsigned int>,
            std::equal_to<unsigned int>,
            unsigned char
            >;

} // namespace mc
