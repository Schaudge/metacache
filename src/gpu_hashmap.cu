#include "hip/hip_runtime.h"
#include <limits>

#include "gpu_hashmap.cuh"
#include "hash_dna.h"
#include "hash_int.h"
#include "sketch_database.h"
#include "gpu_engine.cuh"

namespace mc {

    //---------------------------------------------------------------
    template<>
    sequence_batch<policy::Host>::sequence_batch(size_t maxTargets, size_t maxEncodeLength) :
        maxTargets_{maxTargets}, maxEncodeLength_{maxEncodeLength}, numTargets_{0}
    {
        if(maxTargets_) {
            hipHostMalloc(&targetIds_, maxTargets_*sizeof(target_id));
            hipHostMalloc(&windowOffsets_, maxTargets_*sizeof(window_id));
            hipHostMalloc(&encodeOffsets_, (maxTargets_+1)*sizeof(encodinglen_t));
            encodeOffsets_[0] = 0;
        }
        if(maxEncodeLength_) {
            hipHostMalloc(&encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));
            hipHostMalloc(&encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));
        }
        CUERR
    }
    //---------------------------------------------------------------
    template<>
    sequence_batch<policy::Host>::~sequence_batch() {
        if(maxTargets_) {
            hipHostFree(targetIds_);
            hipHostFree(windowOffsets_);
            hipHostFree(encodeOffsets_);
        }
        if(maxEncodeLength_) {
            hipHostFree(encodedSeq_);
            hipHostFree(encodedAmbig_);
        }
        CUERR
    }

    //---------------------------------------------------------------
    template<>
    sequence_batch<policy::Device>::sequence_batch(size_t maxTargets, size_t maxEncodeLength) :
        maxTargets_{maxTargets}, maxEncodeLength_{maxEncodeLength}, numTargets_{0}
    {
        if(maxTargets_) {
            hipMalloc(&targetIds_, maxTargets_*sizeof(target_id));
            hipMalloc(&windowOffsets_, maxTargets_*sizeof(window_id));
            hipMalloc(&encodeOffsets_, (maxTargets_+1)*sizeof(encodinglen_t));
        }
        if(maxEncodeLength_) {
            hipMalloc(&encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));
            hipMalloc(&encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));
        }
        CUERR
    }
    //---------------------------------------------------------------
    template<>
    sequence_batch<policy::Device>::~sequence_batch() {
        if(maxTargets_) {
            hipFree(targetIds_);
            hipFree(windowOffsets_);
            hipFree(encodeOffsets_);
        }
        if(maxEncodeLength_) {
            hipFree(encodedSeq_);
            hipFree(encodedAmbig_);
        }
        CUERR
    }


    //-----------------------------------------------------
    template<
        class Key,
        class ValueT,
        class Hash,
        class KeyEqual,
        class BucketSizeT
    >
    void gpu_hashmap<Key,ValueT,Hash,KeyEqual,BucketSizeT>::init()
    {
        hipSetDevice(0);
        size_t maxSeqLength = MAX_ENCODE_LENGTH_PER_BATCH*sizeof(encodedambig_t)*CHAR_BIT;
        size_t maxWindows = (maxSeqLength-kmerLength_ + windowStride_) / windowStride_;
        size_t maxFeatures = maxWindows * sketchSize_;

        seqBatchesDevice_.emplace_back(MAX_TARGET_PER_BATCH, MAX_ENCODE_LENGTH_PER_BATCH);

        hipMalloc(&featureBatch_.features_, maxFeatures*sizeof(Key));
        hipMalloc(&featureBatch_.values_, maxFeatures*sizeof(ValueT));
        hipMalloc(&featureBatch_.featureCounter_, sizeof(size_t));
        CUERR
    }


    //-----------------------------------------------------
    template<
        class Key,
        class ValueT,
        class Hash,
        class KeyEqual,
        class BucketSizeT
    >
    std::vector<Key> gpu_hashmap<Key,ValueT,Hash,KeyEqual,BucketSizeT>::insert(
        const sequence_batch<policy::Host>& seqBatchHost
    ) {
        hipStream_t stream = 0;

        seqBatchesDevice_[0].num_targets(seqBatchHost.num_targets());

        //copy batch to gpu
        hipMemcpyAsync(seqBatchesDevice_[0].target_ids(), seqBatchHost.target_ids(),
                        seqBatchHost.num_targets()*sizeof(target_id),
                        hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(seqBatchesDevice_[0].window_offsets(), seqBatchHost.window_offsets(),
                        seqBatchHost.num_targets()*sizeof(window_id),
                        hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(seqBatchesDevice_[0].encode_offsets(), seqBatchHost.encode_offsets(),
                        (seqBatchHost.num_targets()+1)*sizeof(encodinglen_t),
                        hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(seqBatchesDevice_[0].encoded_seq(), seqBatchHost.encoded_seq(),
                        seqBatchHost.encode_offsets()[seqBatchHost.num_targets()]*sizeof(encodedseq_t),
                        hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(seqBatchesDevice_[0].encoded_ambig(), seqBatchHost.encoded_ambig(),
                        seqBatchHost.encode_offsets()[seqBatchHost.num_targets()]*sizeof(encodedambig_t),
                        hipMemcpyHostToDevice, stream);

        //initialize counter
        hipMemsetAsync(featureBatch_.featureCounter_, 0, sizeof(uint64_t), stream);
        // hipStreamSynchronize(stream);
        // CUERR

        // max 32*4 features => max window size is 128
        #define BLOCK_THREADS 32
        #define ITEMS_PER_THREAD 4

        extract_features<BLOCK_THREADS,ITEMS_PER_THREAD><<<1,BLOCK_THREADS,0,stream>>>(
            //todo num targets
            seqBatchesDevice_[0].target_ids(),
            seqBatchesDevice_[0].window_offsets(),
            seqBatchesDevice_[0].encode_offsets(),
            seqBatchesDevice_[0].encoded_seq(),
            seqBatchesDevice_[0].encoded_ambig(),
            kmerLength_, sketchSize_, windowStride_, windowSize_,
            featureBatch_.features_,
            featureBatch_.values_,
            featureBatch_.featureCounter_);
        // hipStreamSynchronize(stream);
        // CUERR

        uint64_t h_featureCounter = 0;
        hipMemcpyAsync(&h_featureCounter, featureBatch_.featureCounter_,
                   sizeof(uint64_t), hipMemcpyDeviceToHost, stream);
        // hipStreamSynchronize(stream);
        // std::cout << "Counter: " << h_featureCounter << '\n';

        // kmer_type * h_features;
        // hipHostMalloc(&h_features, numFeatures*sizeof(Key));
        std::vector<Key> h_features(h_featureCounter);
        std::vector<ValueT> h_values(h_featureCounter);
        hipMemcpyAsync(h_features.data(), featureBatch_.features_,
                   h_featureCounter*sizeof(Key), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(h_values.data(), featureBatch_.values_,
                   h_featureCounter*sizeof(ValueT), hipMemcpyDeviceToHost, stream);
        // hipStreamSynchronize(stream);
        // CUERR

        //print features
        // for(size_t i=0; i<h_featureCounter; ++i) {
        //     std:: cout << h_features[i] << ' ';
        //     if((i+1) % sketchSize_ == 0)
        //         std::cout << '\n';
        // }
        // std::cout << std::endl;

        // for(size_t i=0; i<h_featureCounter; ++i) {
        //     std:: cout << '(' << h_values[i].tgt << ',' << h_values[i].win << ") " ;
        //     if((i+1) % sketchSize_ == 0)
        //         std::cout << '\n';
        // }
        std::cout << std::endl;

        hipStreamSynchronize(stream);
        CUERR

        return h_features;
    }





    //-----------------------------------------------------
    template class gpu_hashmap<
            kmer_type,
            location,
            // uint64_t,
            same_size_hash<kmer_type>,
            std::equal_to<kmer_type>,
            unsigned char
            >;

} // namespace mc
