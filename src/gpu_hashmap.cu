#include "hip/hip_runtime.h"
#include <limits>

#include "gpu_hashmap.cuh"
#include "hash_dna.h"
#include "hash_int.h"
#include "database.h"
#include "gpu_hashmap_operations.cuh"
#include "stat_combined.cuh"

#include "../dep/warpcore/include/single_value_hash_table.cuh"
#include "../dep/warpcore/include/bucket_list_hash_table.cuh"

namespace mc {


template<class SizeT, class BucketSizeT>
__global__
void calculate_sizes_kernel(SizeT * d_offsets, BucketSizeT * d_sizes, SizeT batchSize)
{
    const auto tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < batchSize) {
        d_sizes[tid] = d_offsets[tid+1] - d_offsets[tid];
    }
}


/*************************************************************************//**
 *
 * @brief   key -> values hashed multimap
 *          for building metacache db on GPU
 *
 * @details uses warpcore::MultiValueHashTable to map key -> locations
 *
 * @tparam  Key:    key type
 * @tparam  ValueT: value type
 *
 *****************************************************************************/
template<class Key, class ValueT>
class gpu_hashmap<Key,ValueT>::build_hash_table {

    using key_type   = Key;
    using value_type = ValueT;

    using ranked_lineage = taxonomy::ranked_lineage;

    using hash_table_t = warpcore::BucketListHashTable<
        key_type, value_type,
        // warpcore::defaults::empty_key<key_type>(),       //=0
        key_type(-2),
        warpcore::defaults::tombstone_key<key_type>(),      //=-1
        warpcore::storage::multi_value::BucketListStore<
            value_type,40,bucket_size_bits(),bucket_size_bits()
        >
    >;

    using size_type  = typename hash_table_t::index_type;
    using status_type  = typename warpcore::Status;

public:
    build_hash_table(
        size_type key_capacity,
        size_type value_capacity,
        std::uint64_t maxLocationsPerFeature
    ) :
        hashTable_{key_capacity, value_capacity,
            warpcore::defaults::seed<key_type>(),   // seed
            1.051, 1, max_bucket_size(),            // grow factor, min & max bucket size
            // 1.075, 3, 26,                           // grow factor, min & max bucket size
            maxLocationsPerFeature},                // max values per key
        batchSize_{default_batch_size()},
        seqBatches_{},
        currentSeqBatch_{0}
    {
        std::cerr << "hashtable status: " << hashTable_.pop_status() << "\n";

        seqBatches_.emplace_back(MAX_TARGETS_PER_BATCH, MAX_LENGTH_PER_BATCH);
        seqBatches_.emplace_back(MAX_TARGETS_PER_BATCH, MAX_LENGTH_PER_BATCH);

        hipStreamCreate(&copyStream_); CUERR
        hipStreamCreate(&insertStream_); CUERR
        hipStreamCreate(&statusStream_); CUERR

        // hipDeviceSynchronize(); CUERR
    }

    //---------------------------------------------------------------
    bool validate() {
        if(hashTable_.peek_status(statusStream_) - status_type::max_values_for_key_reached())
            return false;
        return true;
    }

    //---------------------------------------------------------------
    status_type pop_status() {
        return hashTable_.pop_status();
    }

    //---------------------------------------------------------------
    static constexpr size_type default_batch_size() noexcept {
        return size_type(1) << 20;
    }
    //-----------------------------------------------------
    size_type batch_size() const noexcept {
        return batchSize_;
    }
    //---------------------------------------------------------------
    float load_factor() noexcept {
        return hashTable_.storage_density();
    }
    //---------------------------------------------------------------
    size_type bucket_count() const noexcept {
        return hashTable_.key_capacity();
    }
    //-----------------------------------------------------
    size_type key_count() noexcept {
        return hashTable_.num_keys();
    }
    //-----------------------------------------------------
    size_type location_count() noexcept {
        return hashTable_.num_values();
    }

    //---------------------------------------------------------------
    void insert_async(
        sequence_batch<policy::Host>& seqBatchHost,
        const sketcher& targetSketcher
    ) {
        // wait for previous insert of current batch to finish
        hipStreamWaitEvent(copyStream_, seqBatches_[currentSeqBatch_].event(), 0); CUERR

        copy_host_to_device_async(
            seqBatchHost, seqBatches_[currentSeqBatch_], copyStream_);

        hipEventRecord(seqBatchHost.event(), copyStream_); CUERR

        hipStreamWaitEvent(insertStream_, seqBatchHost.event(), 0); CUERR

        constexpr int maxSketchSize = 16;

        // max 32*4 characters per warp, so max window size is 128
        if(targetSketcher.window_size() <= 128 && targetSketcher.sketch_size() <= maxSketchSize) {
            constexpr int warpsPerBlock = 2;
            constexpr int threadsPerBlock = 32*warpsPerBlock;

            const dim3 numBlocks{1024, seqBatches_[currentSeqBatch_].num_targets()};
            insert_features<threadsPerBlock,maxSketchSize>
                <<<numBlocks,threadsPerBlock,0,insertStream_>>>(
                hashTable_,
                seqBatches_[currentSeqBatch_].num_targets(),
                seqBatches_[currentSeqBatch_].target_ids(),
                seqBatches_[currentSeqBatch_].window_offsets(),
                seqBatches_[currentSeqBatch_].sequence(),
                seqBatches_[currentSeqBatch_].sequence_offsets(),
                targetSketcher.kmer_size(),
                targetSketcher.sketch_size(),
                targetSketcher.window_size(),
                targetSketcher.window_stride());
        }
        else {
            std::cerr << "Max window size is 128!\n";
            std::cerr << "Max sketch size is " << maxSketchSize << "\n";
        }

        hipEventRecord(seqBatches_[currentSeqBatch_].event(), insertStream_); CUERR

        // hipStreamSynchronize(insertStream_); CUERR

        currentSeqBatch_ ^= 1;
    }

    //-----------------------------------------------------
    void wait_until_insert_finished() const {
        hipStreamSynchronize(insertStream_); CUERR
    }

    //---------------------------------------------------------------
    statistics_accumulator_gpu<policy::Host>
    location_list_size_statistics()
    {
        hipDeviceSynchronize(); CUERR

        key_type * keys = nullptr;
        size_type numKeys = hashTable_.num_keys(); CUERR
        hipMalloc(&keys, numKeys*sizeof(key_type)); CUERR
        hashTable_.retrieve_all_keys(keys, numKeys); CUERR

        size_type  * numValuesBuffer_d = nullptr;
        hipMalloc(&numValuesBuffer_d, batchSize_*sizeof(size_type)); CUERR

        size_type * valuesCountPtr = nullptr;
        hipHostMalloc(&valuesCountPtr, sizeof(size_type)); CUERR
        *valuesCountPtr = 0;

        statistics_accumulator_gpu<policy::Device> accumulator_d{};

        const size_type numCycles = numKeys / batchSize_;
        const size_type lastBatchSize = numKeys % batchSize_;

        for(size_type b = 0; b < numCycles; ++b) {
            hashTable_.num_values(
                keys+b*batchSize_, batchSize_, *valuesCountPtr,
                numValuesBuffer_d);
            CUERR

            accumulator_d.accumulate(numValuesBuffer_d, batchSize_);
        }
        if(lastBatchSize) {
            hashTable_.num_values(
                keys+numCycles*batchSize_, lastBatchSize, *valuesCountPtr,
                numValuesBuffer_d);
            CUERR

            accumulator_d.accumulate(numValuesBuffer_d, lastBatchSize);
        }

        hipFree(keys); CUERR
        hipFree(numValuesBuffer_d); CUERR
        hipHostFree(valuesCountPtr); CUERR

        statistics_accumulator_gpu<policy::Host> accumulator_h{};
        accumulator_h = accumulator_d;

        return accumulator_h;
    }


private:
    class retrieval_buffer {
    public:
        retrieval_buffer(size_type batchSize) :
            valuesAlloc_{0},
            d_values_{nullptr},
            h_values_{nullptr}
        {
            hipHostMalloc(&valuesCountPtr_, sizeof(size_type)); CUERR
            valuesCountPtr_[0] = 0;

            hipHostMalloc(&h_keys_, batchSize*sizeof(key_type)); CUERR
            hipMalloc    (&d_offsets_, (batchSize+1)*sizeof(size_type)); CUERR
            hipMalloc    (&d_sizes_, batchSize*sizeof(bucket_size_type)); CUERR
            hipHostMalloc(&h_sizes_, batchSize*sizeof(bucket_size_type)); CUERR

            hipStreamCreate(&stream_);
        }

        ~retrieval_buffer() {
            hipHostFree(valuesCountPtr_); CUERR
            hipHostFree(h_keys_); CUERR
            hipFree    (d_offsets_); CUERR
            hipFree    (d_sizes_); CUERR
            hipHostFree(h_sizes_); CUERR
            if(valuesAlloc_) {
                hipFree    (d_values_); CUERR
                hipHostFree(h_values_); CUERR
            }

            hipStreamDestroy(stream_);
        }

        size_type * values_count() const noexcept { return valuesCountPtr_; }
        size_type * d_offsets() const noexcept { return d_offsets_; }
        key_type * h_keys() const noexcept { return h_keys_; }
        bucket_size_type * d_sizes() const noexcept { return d_sizes_; }
        bucket_size_type * h_sizes() const noexcept { return h_sizes_; }
        value_type * d_values() const noexcept { return d_values_; }
        value_type * h_values() const noexcept { return h_values_; }
        hipStream_t stream() const noexcept { return stream_; }

        void resize() {
            if(*values_count() > valuesAlloc_) {
                valuesAlloc_ = *values_count() * 1.1;
                hipHostFree(h_values()); CUERR
                hipFree    (d_values()); CUERR
                hipHostMalloc(&h_values_, valuesAlloc_*sizeof(value_type)); CUERR
                hipMalloc    (&d_values_, valuesAlloc_*sizeof(value_type)); CUERR
            }
        }

    private:
        size_type valuesAlloc_;
        size_type * valuesCountPtr_;

        key_type   * h_keys_;
        size_type  * d_offsets_;
        bucket_size_type * d_sizes_;
        bucket_size_type * h_sizes_;
        value_type * d_values_;
        value_type * h_values_;

        hipStream_t stream_;
    };

    //---------------------------------------------------------------
    void retrieve_and_write_binary(
        std::ostream& os,
        key_type * d_keys,
        retrieval_buffer& buffer,
        size_type batchSize,
        std::mutex& mtx
    ) {
        // get valuesCount
        hashTable_.num_values(
            d_keys, batchSize,
            *(buffer.values_count()),
            buffer.d_offsets()+1,
            buffer.stream());
        hipStreamSynchronize(buffer.stream()); CUERR

        // reallocate if buffers to small
        buffer.resize();

        // get values
        hashTable_.retrieve(
            d_keys, batchSize,
            buffer.d_offsets(), buffer.d_offsets()+1,
            buffer.d_values(), *(buffer.values_count()),
            buffer.stream());

        calculate_sizes_kernel<<<SDIV(batchSize, MAXBLOCKSIZE), MAXBLOCKSIZE, 0, buffer.stream()>>>(
            buffer.d_offsets(), buffer.d_sizes(), batchSize);

        hipMemcpyAsync(buffer.h_keys(), d_keys, batchSize*sizeof(key_type),
            hipMemcpyDeviceToHost, buffer.stream());
        hipMemcpyAsync( buffer.h_sizes(),  buffer.d_sizes(), batchSize*sizeof(bucket_size_type),
            hipMemcpyDeviceToHost, buffer.stream());
        hipMemcpyAsync( buffer.h_values(), buffer. d_values(), *buffer.values_count()*sizeof(value_type),
            hipMemcpyDeviceToHost, buffer.stream());

        hipStreamSynchronize(buffer.stream()); CUERR

        const auto tableStatus = hashTable_.pop_status(buffer.stream());
        if(tableStatus.has_any())
            std::cerr << tableStatus << '\n';

        std::lock_guard<std::mutex> lock(mtx);
        write_binary(os, buffer.h_keys(), batchSize);
        write_binary(os, buffer.h_sizes(), batchSize);
        write_binary(os, buffer.h_values(), *buffer.values_count());
    }

public:
    //---------------------------------------------------------------
    void serialize(std::ostream& os)
    {
        hipDeviceSynchronize(); CUERR

        using len_t = std::uint64_t;

        write_binary(os, len_t(key_count()));
        write_binary(os, len_t(location_count()));
        write_binary(os, len_t(batch_size()));

        size_type numKeys = hashTable_.num_keys(); CUERR
        // allocate buffers
        key_type * d_keys;
        hipMalloc(&d_keys, numKeys*sizeof(key_type)); CUERR
        retrieval_buffer buffer0(batchSize_);
        retrieval_buffer buffer1(batchSize_);
        // get keys
        hashTable_.retrieve_all_keys(d_keys, numKeys); CUERR

        const len_t numCycles = numKeys / batchSize_;
        const len_t lastBatchSize = numKeys % batchSize_;
        std::mutex mtx;

        int gpuId = -1;
        hipGetDevice(&gpuId);

        auto retriever0 = std::async(std::launch::async, [&] {
            hipSetDevice(gpuId);

            for(len_t b = 0; b < numCycles; b+=2) {
                retrieve_and_write_binary(os,
                    d_keys + b * batchSize_,
                    buffer0, batchSize_, mtx);
            }
        });
        auto retriever1 = std::async(std::launch::async, [&] {
            hipSetDevice(gpuId);

            for(len_t b = 1; b < numCycles; b+=2) {
                retrieve_and_write_binary(os,
                    d_keys + b * batchSize_,
                    buffer1, batchSize_, mtx);
            }
        });

        retriever0.get();
        retriever1.get();

        if(lastBatchSize) {
            retrieve_and_write_binary(os,
                d_keys + numCycles * batchSize_,
                buffer0, lastBatchSize, mtx);
        }

        hipFree(d_keys); CUERR
    }

private:
    hash_table_t hashTable_;

    size_type batchSize_;

    size_t maxBatches_;
    std::vector<sequence_batch<policy::Device>> seqBatches_;
    unsigned currentSeqBatch_;

    hipStream_t copyStream_;
    hipStream_t insertStream_;
    hipStream_t statusStream_;
};



/*************************************************************************//**
 *
 * @brief   key -> values hashed multimap
 *          loads metacache db to GPU to enable queries on GPU
 *
 * @details uses warpcore::SingleValueHashTable to map key -> locations pointer & size
 *          locations are stored in separate array
 *
 * @tparam  Key:    key type
 * @tparam  ValueT: value type
 *
 *****************************************************************************/
template<class Key, class ValueT>
class gpu_hashmap<Key,ValueT>::query_hash_table {

    using key_type   = Key;
    using value_type = std::uint64_t;
    using location_type = ValueT;
    using size_type  = size_t;

    using ranked_lineage = taxonomy::ranked_lineage;

    using hash_table_t = warpcore::SingleValueHashTable<
        key_type, value_type,
        // warpcore::defaults::empty_key<key_type>(),       //=0
        key_type(-2),
        warpcore::defaults::tombstone_key<key_type>(),      //=-1
        warpcore::defaults::probing_scheme_t<key_type, 8>,
        // warpcore::storage::key_value::SoAStore<key_type, value_type>>;
        warpcore::storage::key_value::AoSStore<key_type, value_type>>;

public:
    query_hash_table(size_t capacity) :
        hashTable_(capacity),
        numKeys_(0), numLocations_(0),
        locations_(nullptr),
        lineages_(nullptr)
    {}

    //---------------------------------------------------------------
    auto pop_status() {
        return hashTable_.pop_status();
    }

    //---------------------------------------------------------------
    float load_factor() noexcept {
        return hashTable_.load_factor();
    }
    //---------------------------------------------------------------
    size_type bucket_count() const noexcept {
        return hashTable_.capacity();
    }
    //-----------------------------------------------------
    size_type key_count() const noexcept {
        return numKeys_;
    }
    //-----------------------------------------------------
    size_type location_count() const noexcept {
        return numLocations_;
    }
    //---------------------------------------------------------------
    ranked_lineage * lineages() const noexcept {
        return lineages_;
    }

    /*************************************************************************//**
    *
    * @brief   query all windows in batch using one warp per window
    *
    * @details saves sketches to gpu memory in case of multi-gpu query
    *
    *****************************************************************************/
    void query_sequences_async(
        uint32_t numWindows,
        const typename query_batch<location_type>::query_gpu_data& gpuData,
        const sketcher& querySketcher,
        bucket_size_type maxLocationsPerFeature) const
    {
        constexpr int maxSketchSize = 16;

        // max 32*4 characters per warp, so max window size is 128
        if(querySketcher.window_size() <= 128 && querySketcher.sketch_size() <= maxSketchSize) {
            constexpr int warpsPerBlock = 2;
            constexpr int threadsPerBlock = 32*warpsPerBlock;

            const int numBlocks = (numWindows+warpsPerBlock-1) / warpsPerBlock;
            gpu_hahstable_query<threadsPerBlock,maxSketchSize>
                <<<numBlocks,threadsPerBlock,0,gpuData.workStream_>>>(
                hashTable_,
                numWindows,
                gpuData.sequenceOffsets_,
                gpuData.sequences_,
                gpuData.sketches_,
                querySketcher.kmer_size(),
                querySketcher.sketch_size(),
                querySketcher.window_size(),
                querySketcher.window_stride(),
                locations_,
                maxLocationsPerFeature,
                gpuData.queryResults_,
                gpuData.resultCounts_
            );
        }
        else {
            std::cerr << "Max window size is 128!\n";
            std::cerr << "Max sketch size is " << maxSketchSize << "\n";
        }
    }

    /*************************************************************************//**
    *
    * @brief   query sketches of all windows in batch using one warp per window
    *
    *****************************************************************************/
    void query_sketches_async(
        uint32_t numWindows,
        const typename query_batch<location_type>::query_gpu_data& gpuData,
        const sketcher& querySketcher,
        bucket_size_type maxLocationsPerFeature) const
    {
        constexpr int maxSketchSize = 16;

        // max 32*4 characters per warp, so max window size is 128
        if(querySketcher.window_size() <= 128 && querySketcher.sketch_size() <= maxSketchSize) {
            constexpr int warpsPerBlock = 2;
            constexpr int threadsPerBlock = 32*warpsPerBlock;

            const int numBlocks = (numWindows+warpsPerBlock-1) / warpsPerBlock;
            gpu_hahstable_query<threadsPerBlock,maxSketchSize>
                <<<numBlocks,threadsPerBlock,0,gpuData.workStream_>>>(
                hashTable_,
                numWindows,
                gpuData.sketches_,
                querySketcher.kmer_size(),
                querySketcher.sketch_size(),
                querySketcher.window_size(),
                querySketcher.window_stride(),
                locations_,
                maxLocationsPerFeature,
                gpuData.queryResults_,
                gpuData.resultCounts_
            );
        }
        else {
            std::cerr << "Max window size is 128!\n";
            std::cerr << "Max sketch size is " << maxSketchSize << "\n";
        }
    }

private:
    //---------------------------------------------------------------
    template<class LenT, class Status>
    LenT deserialize_batch_of_buckets(
        std::istream& is,
        key_type * h_keyBuffer,
        key_type * d_keyBuffer,
        uint64_t * h_offsetBuffer,
        uint64_t * d_offsetBuffer,
        std::vector<bucket_size_type>& bsizeBuffer,
        LenT batchSize,
        location * valueBuffers[2],
        hipEvent_t events[2],
        LenT valBatchSize,
        location * d_values,
        uint64_t valuesOffset,
        Status * status,
        hipStream_t stream)
    {
        using len_t = LenT;
        using handler_type = warpcore::status_handlers::ReturnStatus;

        const size_type probingLength = hashTable_.capacity();

        auto batchValuesOffset = valuesOffset;

        //load batch
        read_binary(is, h_keyBuffer, batchSize);
        read_binary(is, bsizeBuffer.data(), batchSize);

        for(len_t i = 0; i < batchSize; ++i) {
            //store offset and size together in 64bit
            //default is 56bit offset, 8bit size
            h_offsetBuffer[i] = (valuesOffset << sizeof(bucket_size_type)*CHAR_BIT)
                                + bsizeBuffer[i];

            valuesOffset += bsizeBuffer[i];
        }

        //check status from previous batch
        //implicit sync
        const auto tableStatus = hashTable_.pop_status(stream);
        if(tableStatus.has_any()) {
            std::cerr << tableStatus << '\n';
            for(size_t j=0; j<batchSize; ++j) {
                if(status[j].has_any()) {
                    std::cerr << h_keyBuffer[j] << ' ' << status[j] << '\n';
                }
            }
        }

        //insert batch
        hipMemcpy(d_keyBuffer, h_keyBuffer, batchSize*sizeof(key_type),
                    hipMemcpyHostToDevice);
        hipMemcpy(d_offsetBuffer, h_offsetBuffer, batchSize*sizeof(uint64_t),
                    hipMemcpyHostToDevice);
        // insert(d_keyBuffer, d_offsetBuffer, batchSize);
        hashTable_.template insert<handler_type>(
            d_keyBuffer, d_offsetBuffer, batchSize, stream, probingLength, status);


        std::uint64_t batchValuesCount = valuesOffset - batchValuesOffset;
        //read batches of locations and copy to device
        const len_t numBatches = batchValuesCount / valBatchSize;
        const size_t remainingSize = batchValuesCount % valBatchSize;

        d_values += batchValuesOffset;

        for(len_t i = 0; i < numBatches; ++i) {
            const len_t id = i % 2;
            hipEventSynchronize(events[id]);
            read_binary(is, valueBuffers[id], valBatchSize);
            hipMemcpyAsync(d_values, valueBuffers[id], valBatchSize*sizeof(location),
                            hipMemcpyHostToDevice, stream);
            hipEventRecord(events[id], stream);

            d_values += valBatchSize;
        }
        //read remaining locations and copy to device
        const len_t id = numBatches % 2;
        hipEventSynchronize(events[id]);
        read_binary(is, valueBuffers[id], remainingSize);
        hipMemcpyAsync(d_values, valueBuffers[id], remainingSize*sizeof(location),
                        hipMemcpyHostToDevice, stream);


        return batchValuesCount;
    }

public:
    //---------------------------------------------------------------
    template<class LenT>
    void deserialize(std::istream& is, LenT nkeys, LenT nlocations)
    {
        using len_t = LenT;

        len_t batchSize = 0;
        read_binary(is, batchSize);

        //TODO tune sizes
        const len_t valBatchSize = 1UL << 20;

        hipStream_t stream = 0;

        //allocate large memory chunk for all locations,
        //individual buckets will then point into this array
        hipMalloc(&locations_, nlocations*sizeof(location)); CUERR
        uint64_t locsOffset = 0;

        {//load hash table
            //allocate insert buffers
            key_type * h_keyBuffer;
            key_type * d_keyBuffer;
            hipHostMalloc(&h_keyBuffer, batchSize*sizeof(key_type));
            hipMalloc    (&d_keyBuffer, batchSize*sizeof(key_type));
            uint64_t * h_offsetBuffer;
            uint64_t * d_offsetBuffer;
            hipHostMalloc(&h_offsetBuffer, batchSize*sizeof(uint64_t));
            hipMalloc    (&d_offsetBuffer, batchSize*sizeof(uint64_t));
            location * valueBuffers[2];
            hipHostMalloc(&valueBuffers[0], valBatchSize*sizeof(location));
            hipHostMalloc(&valueBuffers[1], valBatchSize*sizeof(location));
            hipEvent_t events[2];
            hipEventCreate(&events[0]);
            hipEventCreate(&events[1]);
            CUERR

            std::vector<bucket_size_type> bsizeBuffer(batchSize);

            using handler_type = warpcore::status_handlers::ReturnStatus;
            using handler_base_type = handler_type::base_type;

            handler_base_type * status;
            hipMallocManaged(&status, batchSize*sizeof(handler_base_type));
            hipMemset(status, 0, batchSize*sizeof(handler_base_type));

            //load full batches
            const len_t numBatches = nkeys / batchSize;
            for(len_t b = 0; b < numBatches; ++b) {
                auto batchValuesCount = deserialize_batch_of_buckets(is,
                    h_keyBuffer, d_keyBuffer, h_offsetBuffer, d_offsetBuffer,
                    bsizeBuffer, batchSize,
                    valueBuffers, events, valBatchSize, locations_, locsOffset,
                    status, stream);

                locsOffset += batchValuesCount;
            }

            //load last batch
            const size_t remainingSize = nkeys % batchSize;
            if(remainingSize) {
                auto batchValuesCount = deserialize_batch_of_buckets(is,
                    h_keyBuffer, d_keyBuffer, h_offsetBuffer, d_offsetBuffer,
                    bsizeBuffer, remainingSize,
                    valueBuffers, events, valBatchSize, locations_, locsOffset,
                    status, stream);

                locsOffset += batchValuesCount;

                //check status from last batch
                //implicit sync
                const auto tableStatus = hashTable_.pop_status(stream);
                if(tableStatus.has_any()) {
                    std::cerr << tableStatus << '\n';
                    for(size_t j=0; j<batchSize; ++j) {
                        if(status[j].has_any()) {
                            std::cerr << h_keyBuffer[j] << ' ' << status[j] << '\n';
                        }
                    }
                }
            }

            hipHostFree(h_keyBuffer);
            hipFree    (d_keyBuffer);
            hipHostFree(h_offsetBuffer);
            hipFree    (d_offsetBuffer);

            hipHostFree(valueBuffers[0]);
            hipHostFree(valueBuffers[1]);
            hipEventDestroy(events[0]);
            hipEventDestroy(events[1]);
            CUERR
        }

        numKeys_ = nkeys;
        numLocations_ = nlocations;
    }

    //---------------------------------------------------------------
    void copy_target_lineages_to_gpu(const std::vector<ranked_lineage>& lins) {
        const size_t size = lins.size()*sizeof(ranked_lineage);
        hipMalloc(&lineages_, size);
        hipMemcpy(lineages_, lins.data(), size, hipMemcpyHostToDevice);
    }

private:
    hash_table_t hashTable_;

    size_type numKeys_;
    size_type numLocations_;
    location * locations_;

    ranked_lineage * lineages_;
};



//---------------------------------------------------------------
template<class Key, class ValueT>
gpu_hashmap<Key,ValueT>::gpu_hashmap() :
    numGPUs_(0),
    maxLoadFactor_(default_max_load_factor()),
    maxLocationsPerFeature_(max_supported_locations_per_feature()),
    valid_(true)
{
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount); CUERR
    numGPUs_ = deviceCount;

    std::cerr << "found " << numGPUs_ << " CUDA devices\n";
}

//-----------------------------------------------------
template<class Key, class ValueT>
gpu_hashmap<Key,ValueT>::~gpu_hashmap() = default;

//-----------------------------------------------------
template<class Key, class ValueT>
gpu_hashmap<Key,ValueT>::gpu_hashmap(gpu_hashmap&& other) :
    numGPUs_{other.numGPUs_},
    maxLoadFactor_{other.maxLoadFactor_},
    maxLocationsPerFeature_{other.maxLocationsPerFeature_},
    valid_{other.valid_.exchange(false)},
    buildHashTables_{std::move(other.buildHashTables_)},
    queryHashTables_{std::move(other.queryHashTables_)}
{};



//---------------------------------------------------------------
template<class Key, class ValueT>
void gpu_hashmap<Key,ValueT>::pop_status(part_id gpuId) {
    if(gpuId < buildHashTables_.size()) {
        hipSetDevice(gpuId); CUERR
        std::cerr
            << "gpu " << gpuId
            << " hashtable status: " << buildHashTables_[gpuId].pop_status() << "\n";
    }
    else if(gpuId < queryHashTables_.size()) {
        hipSetDevice(gpuId); CUERR
        std::cerr
            << "gpu " << gpuId
            << " hashtable status: " << queryHashTables_[gpuId].pop_status() << "\n";
    }
}
//---------------------------------------------------------------
template<class Key, class ValueT>
void gpu_hashmap<Key,ValueT>::pop_status() {
    for(part_id gpuId = 0; gpuId < buildHashTables_.size(); ++gpuId) {
        hipSetDevice(gpuId); CUERR
        std::cerr
            << "gpu " << gpuId
            <<  " hashtable status: " << buildHashTables_[gpuId].pop_status() << "\n";
    }
    for(part_id gpuId = 0; gpuId < queryHashTables_.size(); ++gpuId) {
        hipSetDevice(gpuId); CUERR
        std::cerr
            << "gpu " << gpuId
            <<  " hashtable status: " << queryHashTables_[gpuId].pop_status() << "\n";
    }
}

//---------------------------------------------------------------
template<class Key, class ValueT>
size_t gpu_hashmap<Key,ValueT>::bucket_count() const noexcept {
    size_t count = 0;
    for(part_id gpuId = 0; gpuId < buildHashTables_.size(); ++gpuId) {
        hipSetDevice(gpuId); CUERR
        count += buildHashTables_[gpuId].bucket_count();
    }
    for(part_id gpuId = 0; gpuId < queryHashTables_.size(); ++gpuId) {
        hipSetDevice(gpuId); CUERR
        count += queryHashTables_[gpuId].bucket_count();
    }
    return count;
}

//---------------------------------------------------------------
template<class Key, class ValueT>
size_t gpu_hashmap<Key,ValueT>::key_count() noexcept {
    size_t count = 0;
    for(part_id gpuId = 0; gpuId < buildHashTables_.size(); ++gpuId) {
        hipSetDevice(gpuId); CUERR
        count += buildHashTables_[gpuId].key_count();
    }
    for(part_id gpuId = 0; gpuId < queryHashTables_.size(); ++gpuId) {
        hipSetDevice(gpuId); CUERR
        count += queryHashTables_[gpuId].key_count();
    }
    return count;
}

//-----------------------------------------------------
template<class Key, class ValueT>
size_t gpu_hashmap<Key,ValueT>::value_count() noexcept {
    size_t count = 0;
    for(part_id gpuId = 0; gpuId < buildHashTables_.size(); ++gpuId) {
        hipSetDevice(gpuId); CUERR
        count += buildHashTables_[gpuId].location_count();
    }
    for(part_id gpuId = 0; gpuId < queryHashTables_.size(); ++gpuId) {
        hipSetDevice(gpuId); CUERR
        count += queryHashTables_[gpuId].location_count();
    }
    return count;
}

//---------------------------------------------------------------
template<class Key, class ValueT>
statistics_accumulator_gpu<policy::Host>
gpu_hashmap<Key,ValueT>::location_list_size_statistics() {
    statistics_accumulator_gpu<policy::Host> totalAccumulator = {};

    for(part_id gpuId = 0; gpuId < numGPUs_; ++gpuId) {
        hipSetDevice(gpuId); CUERR
        auto accumulator = buildHashTables_[gpuId].location_list_size_statistics();

        std::cout
            << "------------------------------------------------\n"
            << "gpu " << gpuId << ":\n"
            << "hashtable status     " << buildHashTables_[gpuId].pop_status() << '\n'
            << "buckets              " << buildHashTables_[gpuId].bucket_count() << '\n'
            << "bucket size          " << "max: " << accumulator.max()
                                       << " mean: " << accumulator.mean()
                                       << " +/- " << accumulator.stddev()
                                       << " <> " << accumulator.skewness() << '\n'
            << "features             " << std::uint64_t(accumulator.size()) << '\n'
            << "dead features        " << dead_feature_count() << '\n'
            << "locations            " << std::uint64_t(accumulator.sum()) << '\n';
            // << "load                 " << buildHashTables_[gpuId].load_factor() << '\n';

        totalAccumulator += accumulator;
    }

    return totalAccumulator;
}



//---------------------------------------------------------------
template<class Key, class ValueT>
part_id gpu_hashmap<Key,ValueT>::initialize_build_hash_tables(part_id numGPUs)
{
    if(numGPUs < numGPUs_)
        numGPUs_ = numGPUs;

    std::cerr << "using " << numGPUs_ << " CUDA devices\n";

    insertBuffers_.reserve(numGPUs_);

    for(part_id gpuId = 0; gpuId < numGPUs_; ++gpuId) {
        hipSetDevice(gpuId); CUERR

        size_t freeMemory = 0;
        size_t totalMemory = 0;
        hipMemGetInfo(&freeMemory, &totalMemory); CUERR
        std::cerr << "gpu " << gpuId << " freeMemory: " << helpers::B2GB(freeMemory) << " GB\n";

        // keep 1 GB of memory free aside from hash table
        const size_t tableMemory = freeMemory - (1ULL << 30);

        constexpr size_t valueSize = sizeof(ValueT);

        const size_t keyCapacity   = tableMemory *  2/13 / (2*valueSize);
        const size_t valueCapacity = tableMemory * 11/13 / valueSize;

        std::cerr << "gpu " << gpuId
                  << " allocate hashtable for " << keyCapacity << " keys"
                                       " and " << valueCapacity << " values\n";
        buildHashTables_.emplace_back(keyCapacity, valueCapacity, maxLocationsPerFeature_);

        hipMemGetInfo(&freeMemory, &totalMemory); CUERR
        std::cerr << "gpu " << gpuId << " freeMemory: " << helpers::B2GB(freeMemory) << " GB\n";

        // allocate host buffers
        insertBuffers_.emplace_back();
    }

    return numGPUs_;
}


//---------------------------------------------------------------
template<class Key, class ValueT>
window_id gpu_hashmap<Key,ValueT>::add_target(
    part_id gpuId, const sequence& seq, target_id tgt, const sketcher& targetSketcher)
{
    using std::begin;
    using std::end;

    return add_target(gpuId, begin(seq), end(seq), tgt, targetSketcher);
}
//-----------------------------------------------------
template<class Key, class ValueT>
window_id gpu_hashmap<Key,ValueT>::add_target(
    part_id gpuId,
    sequence::const_iterator first,
    sequence::const_iterator last,
    target_id tgt,
    const sketcher& targetSketcher)
{
    // std::cerr << "add target " << tgt << " to gpu " << gpuId << "\n";

    using std::distance;

    window_id totalWindows = 0;

    for(window_id processedWindows = 0;
        distance(first, last) >= targetSketcher.kmer_size();
        first += processedWindows*targetSketcher.window_stride())
    {
        //fill sequence batch
        processedWindows = insertBuffers_[gpuId].current_seq_batch().add_target(
            first, last, tgt, totalWindows, targetSketcher);

        // if no windows were processed batch must be full
        if(!processedWindows && insertBuffers_[gpuId].current_seq_batch().num_targets()) {
            // std::cerr << "gpu " << gpuId << " insert\n";
            insert(gpuId, insertBuffers_[gpuId].current_seq_batch(), targetSketcher);
            insertBuffers_[gpuId].switch_seq_batch();
            insertBuffers_[gpuId].current_seq_batch().clear();
        }

        totalWindows += processedWindows;
    }

    return totalWindows;
}


//---------------------------------------------------------------
template<class Key, class ValueT>
void gpu_hashmap<Key,ValueT>::insert(
    part_id gpuId,
    sequence_batch<policy::Host>& seqBatchHost,
    const sketcher& targetSketcher)
{
    hipSetDevice(gpuId); CUERR

    if(valid_ && buildHashTables_[gpuId].validate()) {
        buildHashTables_[gpuId].insert_async(
            seqBatchHost, targetSketcher);
    }
    else {
        valid_ = false;
    }
}
//-----------------------------------------------------
template<class Key, class ValueT>
void gpu_hashmap<Key,ValueT>::wait_until_add_target_complete(
    part_id gpuId, const sketcher& targetSketcher)
{
    if(gpuId < numGPUs_) {
        hipSetDevice(gpuId); CUERR

        if(insertBuffers_[gpuId].current_seq_batch().num_targets()) {
            insert(gpuId, insertBuffers_[gpuId].current_seq_batch(), targetSketcher);
        }

        buildHashTables_[gpuId].wait_until_insert_finished();
    }
}


//---------------------------------------------------------------
template<class Key, class ValueT>
void gpu_hashmap<Key,ValueT>::query_async(
    query_batch<value_type>& batch,
    part_id hostId,
    const sketcher& querySketcher,
    bool copyAllHits,
    taxon_rank lowestRank) const
{
    for(part_id gpuId = 0; gpuId < batch.num_gpus(); ++gpuId)
    {
        hipSetDevice(gpuId); CUERR

        batch.wait_for_allhits_copied(gpuId);

        if(gpuId == 0) {
            batch.copy_queries_to_device_async(hostId);

            queryHashTables_[gpuId].query_sequences_async(
                batch.host_data(hostId).num_windows(),
                batch.gpu_data(gpuId),
                querySketcher,
                maxLocationsPerFeature_);
        }
        else {
            queryHashTables_[gpuId].query_sketches_async(
                batch.host_data(hostId).num_windows(),
                batch.gpu_data(gpuId),
                querySketcher,
                maxLocationsPerFeature_);
        }
        batch.mark_query_finished(gpuId);

        // batch.sync_work_stream(gpuId); CUERR

        if(gpuId < batch.num_gpus()-1)
            batch.copy_queries_to_next_device_async(hostId, gpuId);

        batch.compact_sort_and_copy_allhits_async(hostId, gpuId, copyAllHits);

        batch.generate_and_copy_top_candidates_async(
            hostId, gpuId, queryHashTables_[gpuId].lineages(), lowestRank);

        // batch.sync_copy_stream(gpuId); CUERR
    }
}


//---------------------------------------------------------------
template<class Key, class ValueT>
void gpu_hashmap<Key,ValueT>::deserialize(std::istream& is, part_id gpuId)
{
    using len_t = std::uint64_t;

    len_t nkeys = 0;
    read_binary(is, nkeys);
    len_t nvalues = 0;
    read_binary(is, nvalues);

    std::cerr << "\n\t#features: " << nkeys << " #locations: " << nvalues << "\n";

    if(nkeys > 0) {
        hipSetDevice(gpuId); CUERR
        std::cerr << "\tloading database to gpu " << gpuId << "\n";

        size_t freeMemory = 0;
        size_t totalMemory = 0;
        hipMemGetInfo(&freeMemory, &totalMemory); CUERR
        std::cerr << "\tfreeMemory: " << helpers::B2GB(freeMemory) << " GB\n";

        //initialize hash table
        queryHashTables_.emplace_back(nkeys/maxLoadFactor_);

        std::cerr << "\tfeatures capacity: " << queryHashTables_.back().bucket_count() << "\n";

        size_t indexSize = queryHashTables_.back().bucket_count() * (sizeof(value_type) + sizeof(value_type));
        std::cerr << "\tindex size: " << helpers::B2GB(indexSize) << " GB\n";

        // load hash table
        queryHashTables_.back().deserialize(is, nkeys, nvalues);

        size_t valuesSize = nvalues*sizeof(location);
        std::cerr << "\tlocations size: " << helpers::B2GB(valuesSize) << " GB\n";
        std::cerr << "\ttotal size: " << helpers::B2GB(indexSize + valuesSize) << " GB\n";

        hipMemGetInfo(&freeMemory, &totalMemory); CUERR
        std::cerr << "\tfreeMemory: " << helpers::B2GB(freeMemory) << " GB\n";
    }
}


//---------------------------------------------------------------
/**
* @brief binary serialization of all non-empty buckets
*/
template<class Key, class ValueT>
void gpu_hashmap<Key,ValueT>::serialize(std::ostream& os, part_id gpuId)
{
    hipSetDevice(gpuId); CUERR
    buildHashTables_[gpuId].serialize(os);
}


//---------------------------------------------------------------
template<class Key, class ValueT>
void gpu_hashmap<Key,ValueT>::copy_target_lineages_to_gpu(
    const std::vector<ranked_lineage>& lins,
    part_id gpuId)
{
    hipSetDevice(gpuId); CUERR
    queryHashTables_[gpuId].copy_target_lineages_to_gpu(lins);
}


//---------------------------------------------------------------
template<class Key, class ValueT>
void gpu_hashmap<Key,ValueT>::enable_all_peer_access(part_id numGPUs)
{
    for (part_id srcId = 0; srcId < numGPUs; ++srcId) {
        hipSetDevice(srcId);
        for (part_id dstId = 0; dstId < numGPUs; ++dstId) {
            if (srcId != dstId) {
                 hipDeviceEnablePeerAccess(dstId, 0);
            }
        }
    }
}



//---------------------------------------------------------------
template class gpu_hashmap<kmer_type, location>;

} // namespace mc
