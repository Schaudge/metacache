#include <limits>

#include "gpu_hashmap.cuh"
#include "hash_dna.h"
#include "hash_int.h"
#include "sketch_database.h"
#include "gpu_engine.cuh"

namespace mc {

    //-----------------------------------------------------
    template<
        class Key,
        class ValueT,
        class Hash,
        class KeyEqual,
        class BucketSizeT
    >
    void gpu_hashmap<Key,ValueT,Hash,KeyEqual,BucketSizeT>::init(
        numk_t kmerLength,
        size_t windowStride,
        sketch_size_type sketchSize
    )
    {
        hipSetDevice(0);
        size_t maxTargets = 1;
        size_t maxEncodeLength = 100;
        size_t maxSeqLength = maxEncodeLength*sizeof(encodedambig_t)*CHAR_BIT;
        size_t maxWindows = (maxSeqLength-kmerLength + windowStride) / windowStride;
        size_t maxFeatures = maxWindows * sketchSize;

        hipMalloc(&seqBatch_.targetIds, maxTargets*sizeof(target_id));
        hipMalloc(&seqBatch_.windowOffsets, maxTargets*sizeof(window_id));
        hipMalloc(&seqBatch_.encodeOffsets, (maxTargets+1)*sizeof(uint32_t));

        hipMalloc(&seqBatch_.encodedSeq, maxEncodeLength*sizeof(encodedseq_t));
        hipMalloc(&seqBatch_.encodedAmbig, maxEncodeLength*sizeof(encodedambig_t));

        hipMalloc(&seqBatch_.features, maxFeatures*sizeof(Key));
        hipMalloc(&seqBatch_.values, maxFeatures*sizeof(ValueT));
        hipMalloc(&seqBatch_.featureCounter, sizeof(size_t));
        CUERR
    }


    //-----------------------------------------------------
    template<
        class Key,
        class ValueT,
        class Hash,
        class KeyEqual,
        class BucketSizeT
    >
    std::vector<Key> gpu_hashmap<Key,ValueT,Hash,KeyEqual,BucketSizeT>::insert(
        target_id tgt,
        std::vector<encodedseq_t> encodedSeq,
        std::vector<encodedambig_t> encodedAmbig,
        numk_t k, size_t windowStride, sketch_size_type sketchSize)
    {

        const uint32_t encodedLength = encodedSeq.size();
        //not real seq length but upper bound
        const size_t seqLength = encodedLength*sizeof(encodedambig_t)*CHAR_BIT;
        const size_t numWindows = (seqLength-k + windowStride) / windowStride;
        const size_t numFeatures = numWindows * sketchSize;
        std::cout << "Target ID: " << tgt
                  << " Length: " << seqLength
                  << " Windows: " << numWindows
                  << " Features: " << numFeatures
                  << '\n';

        hipMemset(seqBatch_.featureCounter, 0, sizeof(uint64_t));
        CUERR

        hipMemcpy(seqBatch_.targetIds, &tgt,
                   sizeof(target_id), hipMemcpyHostToDevice);
        const window_id winOffset = 0;
        hipMemcpy(seqBatch_.windowOffsets, &winOffset,
                   sizeof(window_id), hipMemcpyHostToDevice);
        const uint32_t encOffsets[2] = {0,encodedLength};
        hipMemcpy(seqBatch_.encodeOffsets, encOffsets,
                   2*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(seqBatch_.encodedSeq, encodedSeq.data(),
                   encodedLength*sizeof(encodedseq_t), hipMemcpyHostToDevice);
        hipMemcpy(seqBatch_.encodedAmbig, encodedAmbig.data(),
                   encodedLength*sizeof(encodedambig_t), hipMemcpyHostToDevice);
        CUERR

        #define BLOCK_THREADS 32

        extract_features<BLOCK_THREADS,4><<<1,BLOCK_THREADS>>>(
            seqBatch_.targetIds,
            seqBatch_.windowOffsets,
            seqBatch_.encodeOffsets,
            seqBatch_.encodedSeq,
            seqBatch_.encodedAmbig,
            k, windowStride, sketchSize,
            seqBatch_.features,
            seqBatch_.values,
            seqBatch_.featureCounter);
        hipDeviceSynchronize();
        CUERR

        uint64_t h_featureCounter = 0;
        hipMemcpy(&h_featureCounter, seqBatch_.featureCounter,
                   sizeof(uint64_t), hipMemcpyDeviceToHost);
        // std::cout << "Counter: " << h_featureCounter << '\n';

        // kmer_type * h_features;
        // hipHostMalloc(&h_features, numFeatures*sizeof(Key));
        std::vector<Key> h_features(h_featureCounter);
        std::vector<ValueT> h_values(h_featureCounter);
        hipMemcpy(h_features.data(), seqBatch_.features,
                   h_featureCounter*sizeof(Key), hipMemcpyDeviceToHost);
        hipMemcpy(h_values.data(), seqBatch_.values,
                   h_featureCounter*sizeof(ValueT), hipMemcpyDeviceToHost);
        CUERR

        //print features
        // for(size_t i=0; i<h_featureCounter; ++i) {
        //     std:: cout << h_features[i] << ' ';
        //     if((i+1) % sketchSize == 0)
        //         std::cout << '\n';
        // }
        // std::cout << std::endl;

        // for(size_t i=0; i<h_featureCounter; ++i) {
        //     std:: cout << '(' << h_values[i].tgt << ',' << h_values[i].win << ") " ;
        //     if((i+1) % sketchSize == 0)
        //         std::cout << '\n';
        // }
        std::cout << std::endl;

        return h_features;
    }





    //-----------------------------------------------------
    template class gpu_hashmap<
            kmer_type,
            location,
            // uint64_t,
            same_size_hash<kmer_type>,
            std::equal_to<kmer_type>,
            unsigned char
            >;

} // namespace mc
