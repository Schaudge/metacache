#include "hip/hip_runtime.h"
#include <limits>

#include "gpu_hashmap.cuh"
#include "hash_dna.h"
#include "hash_int.h"
#include "sketch_database.h"
#include "gpu_engine.cuh"

namespace mc {

    __global__
    void insert_kernel(target_id tgt, encodedseq_t* encodedSeq, encodedambig_t* encodedAmbig) {

    }


    //-----------------------------------------------------
    template<
        class Key,
        class ValueT,
        class Hash,
        class KeyEqual,
        class BucketSizeT
    >
    void gpu_hashmap<Key,ValueT,Hash,KeyEqual,BucketSizeT>::init()
    {
        hipSetDevice(0);
    }


    //-----------------------------------------------------
    template<
        class Key,
        class ValueT,
        class Hash,
        class KeyEqual,
        class BucketSizeT
    >
    std::vector<Key> gpu_hashmap<Key,ValueT,Hash,KeyEqual,BucketSizeT>::insert(
        target_id tgt,
        std::vector<encodedseq_t> encodedSeq,
        std::vector<encodedambig_t> encodedAmbig,
        size_t seqLength,
        numk_t k, size_t windowStride, sketch_size_type sketchSize)
    {
        encodedseq_t * d_encodedSeq;
        encodedambig_t * d_encodedAmbig;
        size_t encodedLength = encodedSeq.size();

        hipMalloc(&d_encodedSeq, encodedLength*sizeof(encodedseq_t));
        hipMalloc(&d_encodedAmbig, encodedLength*sizeof(encodedambig_t));
        CUERR

        // kmer_type * h_kmers;
        Key * d_kmers;
        ValueT * d_values;
        const size_t numWindows = (seqLength-k + windowStride) / windowStride;
        std::cout << "Target ID: " << tgt << " Length: " << seqLength << " Windows: " << numWindows << '\n';
        const size_t numFeatures = numWindows * sketchSize;
        uint64_t * d_kmerCounter;
        uint64_t h_kmerCounter = 0;

        std::vector<Key> h_kmers(numFeatures);
        std::vector<ValueT> h_values(numFeatures);

        // hipHostMalloc(&h_kmers, numFeatures*sizeof(Key));
        hipMalloc(&d_kmers, numFeatures*sizeof(Key));
        hipMalloc(&d_values, numFeatures*sizeof(ValueT));
        CUERR
        hipMalloc(&d_kmerCounter, sizeof(uint64_t));
        hipMemset(d_kmerCounter, 0, sizeof(uint64_t));
        CUERR

        hipMemcpy(d_encodedSeq, encodedSeq.data(),
                   encodedLength*sizeof(encodedseq_t), hipMemcpyHostToDevice);
        hipMemcpy(d_encodedAmbig, encodedAmbig.data(),
                   encodedLength*sizeof(encodedambig_t), hipMemcpyHostToDevice);
        CUERR

        #define BLOCK_THREADS 32

        window_id winOffset = 0;

        extract_features<BLOCK_THREADS,4><<<1,BLOCK_THREADS>>>(
            tgt, winOffset,
            d_encodedSeq, d_encodedAmbig, seqLength,
            k, windowStride, sketchSize,
            d_kmers,
            d_values,
            d_kmerCounter);
        hipDeviceSynchronize();
        CUERR

        hipMemcpy(&h_kmerCounter, d_kmerCounter, sizeof(uint64_t), hipMemcpyDeviceToHost);
        // std::cout << "Counter: " << h_kmerCounter << '\n';
        h_kmers.resize(h_kmerCounter);
        h_values.resize(h_kmerCounter);
        hipMemcpy(h_kmers.data(), d_kmers, h_kmerCounter*sizeof(Key), hipMemcpyDeviceToHost);
        hipMemcpy(h_values.data(), d_values, h_kmerCounter*sizeof(ValueT), hipMemcpyDeviceToHost);
        CUERR

        //print kmers
        // for(size_t i=0; i<h_kmerCounter; ++i) {
        //     std:: cout << h_kmers[i] << ' ';
        //     if((i+1) % sketchSize == 0)
        //         std::cout << '\n';
        // }
        // std::cout << std::endl;

        // for(size_t i=0; i<h_kmerCounter; ++i) {
        //     std:: cout << '(' << h_values[i].tgt << ',' << h_values[i].win << ") " ;
        //     if((i+1) % sketchSize == 0)
        //         std::cout << '\n';
        // }
        std::cout << std::endl;

        return h_kmers;
    }





    //-----------------------------------------------------
    template class gpu_hashmap<
            kmer_type,
            location,
            // uint64_t,
            same_size_hash<kmer_type>,
            std::equal_to<kmer_type>,
            unsigned char
            >;

} // namespace mc
