
#include "sequence_batch.cuh"

namespace mc {


//---------------------------------------------------------------
template<>
sequence_batch<policy::Host>::sequence_batch(uint32_t maxTargets, size_t maxEncodeLength) :
    maxTargets_{maxTargets}, numTargets_{0}, maxEncodeLength_{maxEncodeLength}
{
    if(maxTargets_) {
        hipHostMalloc(&targetIds_, maxTargets_*sizeof(target_id));
        hipHostMalloc(&windowOffsets_, maxTargets_*sizeof(window_id));
        hipHostMalloc(&encodeOffsets_, (maxTargets_+1)*sizeof(encodinglen_t));
        encodeOffsets_[0] = 0;
    }
    if(maxEncodeLength_) {
        hipHostMalloc(&encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));
        hipHostMalloc(&encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));
    }
    CUERR
}
//---------------------------------------------------------------
template<>
sequence_batch<policy::Host>::~sequence_batch() {
    if(maxTargets_) {
        hipHostFree(targetIds_);
        hipHostFree(windowOffsets_);
        hipHostFree(encodeOffsets_);
    }
    if(maxEncodeLength_) {
        hipHostFree(encodedSeq_);
        hipHostFree(encodedAmbig_);
    }
    CUERR
}

//---------------------------------------------------------------
template<>
sequence_batch<policy::Device>::sequence_batch(uint32_t maxTargets, size_t maxEncodeLength) :
    maxTargets_{maxTargets}, numTargets_{0}, maxEncodeLength_{maxEncodeLength}
{
    if(maxTargets_) {
        hipMalloc(&targetIds_, maxTargets_*sizeof(target_id));
        hipMalloc(&windowOffsets_, maxTargets_*sizeof(window_id));
        hipMalloc(&encodeOffsets_, (maxTargets_+1)*sizeof(encodinglen_t));
    }
    if(maxEncodeLength_) {
        hipMalloc(&encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));
        hipMalloc(&encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));
    }
    CUERR

    size_t totalSize = maxTargets_*(sizeof(target_id) + sizeof(window_id)) +
                       (maxTargets_+1)*sizeof(encodinglen_t) +
                       maxEncodeLength_*(sizeof(encodedseq_t) + sizeof(encodedambig_t));
    std::cerr << "total batch size: " << (totalSize >> 10) << " KB\n";
}
//---------------------------------------------------------------
template<>
sequence_batch<policy::Device>::~sequence_batch() {
    if(maxTargets_) {
        hipFree(targetIds_);
        hipFree(windowOffsets_);
        hipFree(encodeOffsets_);
    }
    if(maxEncodeLength_) {
        hipFree(encodedSeq_);
        hipFree(encodedAmbig_);
    }
    CUERR
}


} // namespace mc
