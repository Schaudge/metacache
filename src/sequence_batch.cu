/******************************************************************************
 *
 * MetaCache - Meta-Genomic Classification Tool
 *
 * Copyright (C) 2016-2021 Robin Kobus  (kobus@uni-mainz.de)
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 *****************************************************************************/

#include "sequence_batch.cuh"

namespace mc {


//---------------------------------------------------------------
template<>
sequence_batch<policy::Host>::sequence_batch(index_type maxTargets, size_type maxSequenceLength) :
    maxTargets_{maxTargets}, numTargets_{0}, maxSequenceLength_{maxSequenceLength}
{
    if (maxTargets_) {
        hipHostMalloc(&targetIds_, maxTargets_*sizeof(target_id));
        hipHostMalloc(&windowOffsets_, maxTargets_*sizeof(window_id));
        hipHostMalloc(&sequenceOffsets_, (maxTargets_+1)*sizeof(size_type));
        sequenceOffsets_[0] = 0;
    }
    if (maxSequenceLength_) {
        hipHostMalloc(&sequence_, maxSequenceLength_*sizeof(char));
    }
    CUERR

    hipEventCreate(&batchProcessedEvent_); CUERR
}
//---------------------------------------------------------------
template<>
sequence_batch<policy::Host>::~sequence_batch() {
    if (maxTargets_) {
        hipHostFree(targetIds_);
        hipHostFree(windowOffsets_);
        hipHostFree(sequenceOffsets_);
    }
    if (maxSequenceLength_) {
        hipHostFree(sequence_);
    }
    CUERR
}

//---------------------------------------------------------------
template<>
sequence_batch<policy::Device>::sequence_batch(index_type maxTargets, size_type maxSequenceLength) :
    maxTargets_{maxTargets}, numTargets_{0}, maxSequenceLength_{maxSequenceLength}
{
    if (maxTargets_) {
        hipMalloc(&targetIds_, maxTargets_*sizeof(target_id));
        hipMalloc(&windowOffsets_, maxTargets_*sizeof(window_id));
        hipMalloc(&sequenceOffsets_, (maxTargets_+1)*sizeof(size_type));
    }
    if (maxSequenceLength_) {
        hipMalloc(&sequence_, maxSequenceLength_*sizeof(char));
    }
    CUERR

    size_t totalSize = maxTargets_*(sizeof(target_id) + sizeof(window_id)) +
                       (maxTargets_+1)*sizeof(size_type) +
                       maxSequenceLength_*sizeof(char);
    std::cerr << "total batch size: " << (totalSize >> 10) << " KB\n";

    hipEventCreate(&batchProcessedEvent_); CUERR
}
//---------------------------------------------------------------
template<>
sequence_batch<policy::Device>::~sequence_batch() {
    if (maxTargets_) {
        hipFree(targetIds_);
        hipFree(windowOffsets_);
        hipFree(sequenceOffsets_);
    }
    if (maxSequenceLength_) {
        hipFree(sequence_);
    }
    CUERR
}


//---------------------------------------------------------------
template<>
void sequence_batch<policy::Host>::clear() noexcept {
    hipEventSynchronize(batchProcessedEvent_); CUERR
    num_targets(0);
}
//-----------------------------------------------------
template<>
void sequence_batch<policy::Device>::clear() noexcept {
    hipEventSynchronize(batchProcessedEvent_); CUERR
    num_targets(0);
}



void copy_host_to_device_async(
    const sequence_batch<policy::Host>& hostBatch,
    sequence_batch<policy::Device>& deviceBatch,
    hipStream_t stream)
{
    using size_type = sequence_batch<policy::Host>::size_type;

    deviceBatch.num_targets(hostBatch.num_targets());

    hipMemcpyAsync(deviceBatch.target_ids(), hostBatch.target_ids(),
                    hostBatch.num_targets()*sizeof(target_id),
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(deviceBatch.window_offsets(), hostBatch.window_offsets(),
                    hostBatch.num_targets()*sizeof(window_id),
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(deviceBatch.sequence_offsets(), hostBatch.sequence_offsets(),
                    (hostBatch.num_targets()+1)*sizeof(size_type),
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(deviceBatch.sequence(), hostBatch.sequence(),
                    hostBatch.sequence_length()*sizeof(char),
                    hipMemcpyHostToDevice, stream);

    // hipStreamSynchronize(stream);
    // CUERR
}


} // namespace mc
