
#include "sequence_batch.cuh"

namespace mc {


//---------------------------------------------------------------
template<>
sequence_batch<policy::Host>::sequence_batch(index_type maxTargets, size_type maxSequenceLength) :
    maxTargets_{maxTargets}, numTargets_{0}, maxSequenceLength_{maxSequenceLength}
{
    if(maxTargets_) {
        hipHostMalloc(&targetIds_, maxTargets_*sizeof(target_id));
        hipHostMalloc(&windowOffsets_, maxTargets_*sizeof(window_id));
        hipHostMalloc(&sequenceOffsets_, (maxTargets_+1)*sizeof(size_type));
        sequenceOffsets_[0] = 0;
    }
    if(maxSequenceLength_) {
        hipHostMalloc(&sequence_, maxSequenceLength_*sizeof(char));
    }
    CUERR

    hipEventCreate(&batchProcessedEvent_); CUERR
}
//---------------------------------------------------------------
template<>
sequence_batch<policy::Host>::~sequence_batch() {
    if(maxTargets_) {
        hipHostFree(targetIds_);
        hipHostFree(windowOffsets_);
        hipHostFree(sequenceOffsets_);
    }
    if(maxSequenceLength_) {
        hipHostFree(sequence_);
    }
    CUERR
}

//---------------------------------------------------------------
template<>
sequence_batch<policy::Device>::sequence_batch(index_type maxTargets, size_type maxSequenceLength) :
    maxTargets_{maxTargets}, numTargets_{0}, maxSequenceLength_{maxSequenceLength}
{
    if(maxTargets_) {
        hipMalloc(&targetIds_, maxTargets_*sizeof(target_id));
        hipMalloc(&windowOffsets_, maxTargets_*sizeof(window_id));
        hipMalloc(&sequenceOffsets_, (maxTargets_+1)*sizeof(size_type));
    }
    if(maxSequenceLength_) {
        hipMalloc(&sequence_, maxSequenceLength_*sizeof(char));
    }
    CUERR

    size_t totalSize = maxTargets_*(sizeof(target_id) + sizeof(window_id)) +
                       (maxTargets_+1)*sizeof(size_type) +
                       maxSequenceLength_*sizeof(char);
    std::cerr << "total batch size: " << (totalSize >> 10) << " KB\n";

    hipEventCreate(&batchProcessedEvent_); CUERR
}
//---------------------------------------------------------------
template<>
sequence_batch<policy::Device>::~sequence_batch() {
    if(maxTargets_) {
        hipFree(targetIds_);
        hipFree(windowOffsets_);
        hipFree(sequenceOffsets_);
    }
    if(maxSequenceLength_) {
        hipFree(sequence_);
    }
    CUERR
}


//---------------------------------------------------------------
template<>
void sequence_batch<policy::Host>::clear() noexcept {
    hipEventSynchronize(batchProcessedEvent_); CUERR
    num_targets(0);
}
//-----------------------------------------------------
template<>
void sequence_batch<policy::Device>::clear() noexcept {
    hipEventSynchronize(batchProcessedEvent_); CUERR
    num_targets(0);
}



void copy_host_to_device_async(
    const sequence_batch<policy::Host>& hostBatch,
    sequence_batch<policy::Device>& deviceBatch,
    hipStream_t stream)
{
    using size_type = sequence_batch<policy::Host>::size_type;

    deviceBatch.num_targets(hostBatch.num_targets());

    hipMemcpyAsync(deviceBatch.target_ids(), hostBatch.target_ids(),
                    hostBatch.num_targets()*sizeof(target_id),
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(deviceBatch.window_offsets(), hostBatch.window_offsets(),
                    hostBatch.num_targets()*sizeof(window_id),
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(deviceBatch.sequence_offsets(), hostBatch.sequence_offsets(),
                    (hostBatch.num_targets()+1)*sizeof(size_type),
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(deviceBatch.sequence(), hostBatch.sequence(),
                    hostBatch.sequence_length()*sizeof(char),
                    hipMemcpyHostToDevice, stream);

    // hipStreamSynchronize(stream);
    // CUERR
}


} // namespace mc
