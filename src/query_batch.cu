#include "hip/hip_runtime.h"

#include "query_batch.cuh"
#include "sketch_database.h"
#include "gpu_result_processing.cuh"

#include "../dep/cub/cub/device/device_segmented_radix_sort.cuh"
#include "../dep/cub/cub/device/device_scan.cuh"

namespace mc {


//---------------------------------------------------------------
template<class result_type>
query_batch<result_type>::query_batch(
    id_type maxQueries,
    size_t maxEncodeLength,
    size_t maxResultsPerQuery,
    uint32_t maxCandidatesPerQuery
) :
    h_numSegments_{0},
    d_numSegments_{0},
    h_numQueries_{0},
    d_numQueries_{0},
    maxQueries_{maxQueries},
    maxEncodeLength_{maxEncodeLength},
    maxResultsPerQuery_{maxResultsPerQuery},
    maxCandidatesPerQuery_{maxCandidatesPerQuery}
{
    //TODO reuse/combine device arrays:
    // d_encodeOffsets_ + d_resultOffsets_

    if(maxQueries_ && maxEncodeLength_ && maxResultsPerQuery_) {
        hipHostMalloc(&h_queryIds_, maxQueries_*sizeof(id_type));
        hipMalloc    (&d_queryIds_, maxQueries_*sizeof(id_type));

        hipHostMalloc(&h_encodeOffsets_, (maxQueries_+1)*sizeof(encodinglen_t));
        hipMalloc    (&d_encodeOffsets_, (maxQueries_+1)*sizeof(encodinglen_t));
        h_encodeOffsets_[0] = 0;

        hipHostMalloc(&h_encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));
        hipMalloc    (&d_encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));

        hipHostMalloc(&h_encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));
        hipMalloc    (&d_encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));

        hipHostMalloc(&h_queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));
        hipMalloc    (&d_queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));

        hipMalloc    (&d_queryResultsTmp_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));

        hipHostMalloc(&h_resultOffsets_, (maxQueries_+1)*sizeof(int));
        hipMalloc    (&d_resultOffsets_, (maxQueries_+1)*sizeof(int));
        h_resultOffsets_[0] = 0;
        hipMemcpy(d_resultOffsets_, h_resultOffsets_, sizeof(int), hipMemcpyHostToDevice);

        hipMalloc    (&d_resultCounts_, maxQueries_*sizeof(int));

        hipHostMalloc(&h_topCandidates_, maxQueries_*maxCandidatesPerQuery_*sizeof(match_candidate));
        hipMalloc    (&d_topCandidates_, maxQueries_*maxCandidatesPerQuery_*sizeof(match_candidate));

        hipHostMalloc(&h_maxWindowsInRange_, maxQueries_*sizeof(window_id));
        hipMalloc    (&d_maxWindowsInRange_, maxQueries_*sizeof(window_id));
    }
    CUERR

    hipStreamCreate(&stream_);
    hipStreamCreate(&resultCopyStream_);

    hipEventCreate(&queriesCopiedEvent_);
    hipEventCreate(&offsetsCopiedEvent_);
    hipEventCreate(&resultReadyEvent_);
    CUERR
}
//---------------------------------------------------------------
template<class result_type>
query_batch<result_type>::~query_batch() {
    CUERR

    if(maxQueries_ && maxEncodeLength_ && maxResultsPerQuery_) {
        hipHostFree(h_queryIds_);
        hipFree    (d_queryIds_);

        hipHostFree(h_encodeOffsets_);
        hipFree    (d_encodeOffsets_);

        hipHostFree(h_encodedSeq_);
        hipFree    (d_encodedSeq_);

        hipHostFree(h_encodedAmbig_);
        hipFree    (d_encodedAmbig_);

        hipHostFree(h_queryResults_);
        hipFree    (d_queryResults_);

        hipFree    (d_queryResultsTmp_);

        hipHostFree(h_resultOffsets_);
        hipFree    (d_resultOffsets_);

        hipFree    (d_resultCounts_);

        hipHostFree(h_topCandidates_);
        hipFree    (d_topCandidates_);

        hipHostFree(h_maxWindowsInRange_);
        hipFree    (d_maxWindowsInRange_);
    }
    CUERR

    hipStreamDestroy(stream_);
    hipStreamDestroy(resultCopyStream_);

    hipEventDestroy(queriesCopiedEvent_);
    hipEventDestroy(offsetsCopiedEvent_);
    hipEventDestroy(resultReadyEvent_);
    CUERR
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::copy_queries_to_device_async() {
    d_numQueries_ = h_numQueries_;
    d_numSegments_ = h_numSegments_;

    hipMemcpyAsync(d_queryIds_, h_queryIds_,
                    d_numQueries_*sizeof(id_type),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_encodeOffsets_, h_encodeOffsets_,
                    (d_numQueries_+1)*sizeof(encodinglen_t),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_encodedSeq_, h_encodedSeq_,
                    h_encodeOffsets_[d_numQueries_]*sizeof(encodedseq_t),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_encodedAmbig_, h_encodedAmbig_,
                    h_encodeOffsets_[d_numQueries_]*sizeof(encodedambig_t),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_maxWindowsInRange_, h_maxWindowsInRange_,
                    d_numSegments_*sizeof(window_id),
                    hipMemcpyHostToDevice, stream_);

    hipEventRecord(queriesCopiedEvent_, stream_);

    // hipStreamSynchronize(stream_);
    // CUERR
}

//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::wait_for_queries_copied() {
    hipEventSynchronize(queriesCopiedEvent_);
}



//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::sync_streams() {
    hipStreamSynchronize(stream_);
    hipStreamSynchronize(resultCopyStream_);
}

//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::sync_result_stream() {
    hipStreamSynchronize(resultCopyStream_);
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::compact_sort_and_copy_results_async() {

    {
        size_t tempStorageBytes = maxQueries_*maxResultsPerQuery_*sizeof(result_type);
        void * d_tempStorage = (void*)(d_queryResultsTmp_);

        // std::cout << "temp size: " << tempStorageBytes << std::endl;

        int numItems = d_numQueries_;
        // size_t tempStorageBytes = 767;
        hipError_t err = hipcub::DeviceScan::InclusiveSum(
            d_tempStorage, tempStorageBytes,
            d_resultCounts_, d_resultCounts_,
            numItems,
            stream_
        );

        // hipStreamSynchronize(stream_);
        // CUERR

        if (err != hipSuccess) {                       \
            std::cout << "CUDA error: " << hipGetErrorString(err) << " : "    \
                    << __FILE__ << ", line " << __LINE__ << std::endl;       \
            exit(1);                                                           \
        }

        compact_kernel<<<d_numQueries_,32,0,stream_>>>(
            d_numQueries_,
            d_resultCounts_,
            maxResultsPerQuery_,
            d_queryResults_,
            d_queryResultsTmp_,
            d_queryIds_,
            d_resultOffsets_
        );

        hipEventRecord(resultReadyEvent_, stream_);
        hipStreamWaitEvent(resultCopyStream_, resultReadyEvent_, 0);

        hipMemcpyAsync(h_resultOffsets_, d_resultOffsets_,
                        (d_numSegments_+1)*sizeof(int),
                        hipMemcpyDeviceToHost, resultCopyStream_);

        hipEventRecord(offsetsCopiedEvent_, resultCopyStream_);

        // hipStreamSynchronize(stream_);
        // CUERR
    }

    using result_type_equivalent = uint64_t;

    static_assert(sizeof(result_type) == sizeof(result_type_equivalent), "result_type must be 64 bit");

    hipcub::DoubleBuffer<result_type_equivalent> d_keys(
        (result_type_equivalent*)(d_queryResultsTmp_),
        (result_type_equivalent*)(d_queryResults_));

    size_t tempStorageBytes = maxEncodeLength_*sizeof(encodedseq_t);
    void * d_tempStorage = (void*)(d_encodedSeq_);

    int numItems = d_numQueries_*maxResultsPerQuery_;
    // int numItems = h_resultOffsets_[d_numSegments_+1];
    // size_t tempStorageBytes = 255;
    hipError_t err = hipcub::DeviceSegmentedRadixSort::SortKeys(
        d_tempStorage, tempStorageBytes,
        d_keys,
        numItems, d_numSegments_,
        d_resultOffsets_, d_resultOffsets_ + 1,
        0, sizeof(result_type_equivalent) * CHAR_BIT,
        stream_
    );

    // hipStreamSynchronize(stream_);
    // CUERR

    if (err != hipSuccess) {                       \
        std::cout << "CUDA error: " << hipGetErrorString(err) << " : "    \
                  << __FILE__ << ", line " << __LINE__ << std::endl;       \
        exit(1);                                                           \
    }

    d_queryResults_    = (result_type*)d_keys.Current();
    d_queryResultsTmp_ = (result_type*)d_keys.Alternate();


    hipEventRecord(resultReadyEvent_, stream_);
    hipStreamWaitEvent(resultCopyStream_, resultReadyEvent_, 0);

    hipEventSynchronize(offsetsCopiedEvent_);

    hipMemcpyAsync(h_queryResults_, d_queryResults_,
                    h_resultOffsets_[d_numSegments_]*sizeof(result_type),
                    hipMemcpyDeviceToHost, resultCopyStream_);

    // hipStreamSynchronize(stream_);
    // CUERR
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::generate_and_copy_top_candidates_async(
    const ranked_lineage * lineages,
    taxon_rank lowestRank)
{
    const size_t numBlocks = d_numSegments_;

    //TODO different max cand cases
    if(maxCandidatesPerQuery_ <= 2) {
        constexpr int maxCandidates = 2;

        generate_top_candidates<maxCandidates><<<numBlocks,32,0,stream_>>>(
            d_numSegments_,
            d_resultOffsets_,
            d_queryResults_,
            d_maxWindowsInRange_,
            lineages,
            lowestRank,
            maxCandidatesPerQuery_,
            d_topCandidates_);

        // hipStreamSynchronize(stream_);
        // CUERR
    }

    hipEventRecord(resultReadyEvent_, stream_);
    hipStreamWaitEvent(resultCopyStream_, resultReadyEvent_, 0);

    // copy candidates to host
    hipMemcpyAsync(h_topCandidates_, d_topCandidates_,
                    d_numSegments_*maxCandidatesPerQuery_*sizeof(match_candidate),
                    hipMemcpyDeviceToHost, resultCopyStream_);

    // hipStreamSynchronize(stream_);
    // CUERR
}


//---------------------------------------------------------------
template class query_batch<location>;

} // namespace mc
