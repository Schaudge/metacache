#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * MetaCache - Meta-Genomic Classification Tool
 *
 * Copyright (C) 2016-2020 Robin Kobus  (kobus@uni-mainz.de)
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 *****************************************************************************/

#include "query_batch.cuh"
#include "database.h"
#include "gpu_result_processing.cuh"

#include <cub/device/device_scan.cuh>

#include "../dep/bb_segsort/src/bb_segsort_keys.cuh"

namespace mc {


//---------------------------------------------------------------
template<class Location>
class query_batch<Location>::segmented_sort
{
    using location_type_equivalent = uint64_t;

    static_assert(sizeof(location_type) == sizeof(location_type_equivalent), "location_type must be 64 bit");

public:
    segmented_sort(
        location_type_equivalent *d_keys, location_type_equivalent *d_keysB,
        const int *d_segs,
        int *d_binnedSegIds,
        int *d_segBinCounters,
        hipStream_t stream)
    :
        sorter_{d_keys, d_keysB,
            d_segs, d_binnedSegIds, d_segBinCounters,
            stream}
    {}

    void run(int numSegs, int maxSegmentSize, hipStream_t stream) const {
        sorter_.run(numSegs, maxSegmentSize, stream);
    }

private:
    bb_segsort_keys<location_type_equivalent, int> sorter_;
};


//---------------------------------------------------------------
template<class Location>
query_batch<Location>::query_host_data::query_host_data(
    index_type maxQueries,
    size_type maxSequenceLength,
    size_type maxResultsPerWindow,
    size_type maxCandidatesPerQuery,
    bool copyAllHits
) :
    numQueries_{0},
    numWindows_{0},
    largestSegmentSize_{0},
    maxCandidatesPerQuery_{maxCandidatesPerQuery}
{
    hipHostMalloc(&queryIds_, maxQueries*sizeof(index_type));
    hipHostMalloc(&sequenceOffsets_, (maxQueries+1)*sizeof(size_type));
    sequenceOffsets_[0] = 0;
    hipHostMalloc(&sequences_, maxSequenceLength*sizeof(char));
    hipHostMalloc(&maxWindowsInRange_, maxQueries*sizeof(window_id));
    CUERR
    if(copyAllHits)
        hipHostMalloc(&queryResults_, maxQueries*maxResultsPerWindow*sizeof(location_type));
    else
        queryResults_ = nullptr;
    hipHostMalloc(&resultOffsets_, (maxQueries+1)*sizeof(int));
    resultOffsets_[0] = 0;
    hipHostMalloc(&topCandidates_, maxQueries*maxCandidatesPerQuery_*sizeof(match_candidate));
    CUERR

    hipEventCreate(&resultsCopiedEvent_);
    CUERR
}
//---------------------------------------------------------------
template<class Location>
query_batch<Location>::query_host_data::~query_host_data()
{
    if(queryIds_)          hipHostFree(queryIds_);
    if(sequenceOffsets_)   hipHostFree(sequenceOffsets_);
    if(sequences_)         hipHostFree(sequences_);
    if(maxWindowsInRange_) hipHostFree(maxWindowsInRange_);
    CUERR
    if(queryResults_)  hipHostFree(queryResults_);
    if(resultOffsets_) hipHostFree(resultOffsets_);
    if(topCandidates_) hipHostFree(topCandidates_);
    CUERR

    if(resultsCopiedEvent_)  hipEventDestroy(resultsCopiedEvent_);
    CUERR
}
//---------------------------------------------------------------
template<class Location>
query_batch<Location>::query_host_data::query_host_data(query_host_data&& other)
{
    numQueries_ = other.numQueries_;
    numWindows_  = other.numWindows_;
    largestSegmentSize_ = other.largestSegmentSize_;
    maxCandidatesPerQuery_  = other.maxCandidatesPerQuery_;

    queryIds_       = other.queryIds_;
    other.queryIds_ = nullptr;
    sequenceOffsets_       = other.sequenceOffsets_;
    other.sequenceOffsets_ = nullptr;
    sequences_       = other.sequences_;
    other.sequences_ = nullptr;
    maxWindowsInRange_       = other.maxWindowsInRange_;
    other.maxWindowsInRange_ = nullptr;

    queryResults_       = other.queryResults_;
    other.queryResults_ = nullptr;
    resultOffsets_       = other.resultOffsets_;
    other.resultOffsets_ = nullptr;
    topCandidates_       = other.topCandidates_;
    other.topCandidates_ = nullptr;

    resultsCopiedEvent_ = other.resultsCopiedEvent_;
    other.resultsCopiedEvent_ = 0;
}
//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::query_host_data::wait_for_results()
{
    hipEventSynchronize(resultsCopiedEvent_);
    CUERR
}



//---------------------------------------------------------------
template<class Location>
query_batch<Location>::query_gpu_data::query_gpu_data(
    index_type maxQueries,
    size_type maxSequenceLength,
    size_type maxSketchSize,
    size_type maxResultsPerWindow,
    size_type maxCandidatesPerQuery,
    bool multiGPU,
    part_id gpuId
)
{
    size_t allocatedGpuMem = 0;

    hipSetDevice(gpuId); CUERR

    hipMalloc    (&queryIds_, maxQueries*sizeof(index_type));
    allocatedGpuMem += maxQueries*sizeof(index_type);
    if(gpuId == 0) {
        // only first gpu holds sequences
        // other gpus use sketches generated by first gpu
        hipMalloc    (&sequenceOffsets_, (maxQueries+1)*sizeof(size_type));
        allocatedGpuMem += (maxQueries+1)*sizeof(size_type);
        hipMalloc    (&sequences_, maxSequenceLength*sizeof(char));
        allocatedGpuMem += maxSequenceLength*sizeof(char);
    }
    else {
        sequenceOffsets_ = nullptr;
        sequences_ = nullptr;
    }

    if(multiGPU) {
        hipMalloc    (&sketches_, maxQueries*maxSketchSize*sizeof(feature_type));
        allocatedGpuMem += maxQueries*maxSketchSize*sizeof(feature_type);
    }
    else {
        sketches_ = nullptr;
    }
    hipMalloc    (&queryResults_, maxQueries*maxResultsPerWindow*sizeof(location_type));
    hipMalloc    (&queryResultsTmp_, maxQueries*maxResultsPerWindow*sizeof(location_type));
    allocatedGpuMem += 2*maxQueries*maxResultsPerWindow*sizeof(location_type);
    hipMalloc    (&resultOffsets_, (maxQueries+1)*sizeof(int));
    allocatedGpuMem += (maxQueries+1)*sizeof(int);
    hipMemset(resultOffsets_, 0, sizeof(int));
    hipMalloc    (&resultCounts_, maxQueries*sizeof(int));
    allocatedGpuMem += maxQueries*sizeof(int);
    hipMalloc    (&segBinCounters_, (SEGBIN_NUM+1)*sizeof(int));
    allocatedGpuMem += (SEGBIN_NUM+1)*sizeof(int);
    hipMalloc    (&topCandidates_, maxQueries*maxCandidatesPerQuery*sizeof(match_candidate));
    allocatedGpuMem += maxQueries*maxCandidatesPerQuery*sizeof(match_candidate);
    hipMalloc    (&maxWindowsInRange_, maxQueries*sizeof(window_id));
    allocatedGpuMem += maxQueries*sizeof(window_id);
    CUERR

    // std::cerr << "query batch size on gpu: " << (allocatedGpuMem >> 20) << " MB\n";

    hipStreamCreate(&workStream_);
    hipStreamCreate(&copyStream_);
    CUERR
    hipEventCreate(&queryFinishedEvent_);
    hipEventCreate(&sketchesCopiedEvent_);
    hipEventCreate(&queryIdsCopiedEvent_);
    hipEventCreate(&queryIdsFinishedEvent_);
    hipEventCreate(&offsetsReadyEvent_);
    hipEventCreate(&offsetsCopiedEvent_);
    hipEventCreate(&allhitsReadyEvent_);
    hipEventCreate(&allhitsCopiedEvent_);
    hipEventCreate(&tophitsReadyEvent_);
    hipEventCreate(&tophitsCopiedEvent_);
    CUERR
}
//---------------------------------------------------------------
template<class Location>
query_batch<Location>::query_gpu_data::query_gpu_data(query_gpu_data&& other)
{
    queryIds_ = other.queryIds_;
    other.queryIds_ = nullptr;
    sequenceOffsets_ = other.sequenceOffsets_;
    other.sequenceOffsets_ = nullptr;
    sequences_ = other.sequences_;
    other.sequences_ = nullptr;
    sketches_ = other.sketches_;
    other.sketches_ = nullptr;
    queryResults_ = other.queryResults_;
    other.queryResults_ = nullptr;
    queryResultsTmp_ = other.queryResultsTmp_;
    other.queryResultsTmp_ = nullptr;
    resultOffsets_ = other.resultOffsets_;
    other.resultOffsets_ = nullptr;
    resultCounts_ = other.resultCounts_;
    other.resultCounts_ = nullptr;
    segBinCounters_ = other.segBinCounters_;
    other.segBinCounters_ = nullptr;
    topCandidates_ = other.topCandidates_;
    other.topCandidates_ = nullptr;
    maxWindowsInRange_ = other.maxWindowsInRange_;
    other.maxWindowsInRange_ = nullptr;

    workStream_ = other.workStream_;
    other.workStream_ = 0;
    copyStream_ = other.copyStream_;
    other.copyStream_ = 0;

    queryFinishedEvent_ = other.queryFinishedEvent_;
    other.queryFinishedEvent_ = 0;
    sketchesCopiedEvent_ = other.sketchesCopiedEvent_;
    other.sketchesCopiedEvent_ = 0;
    queryIdsCopiedEvent_ = other.queryIdsCopiedEvent_;
    other.queryIdsCopiedEvent_ = 0;
    queryIdsFinishedEvent_ = other.queryIdsFinishedEvent_;
    other.queryIdsFinishedEvent_ = 0;
    offsetsReadyEvent_ = other.offsetsReadyEvent_;
    other.offsetsReadyEvent_ = 0;
    offsetsCopiedEvent_ = other.offsetsCopiedEvent_;
    other.offsetsCopiedEvent_ = 0;
    allhitsReadyEvent_ = other.allhitsReadyEvent_;
    other.allhitsReadyEvent_ = 0;
    allhitsCopiedEvent_ = other.allhitsCopiedEvent_;
    other.allhitsCopiedEvent_ = 0;
    tophitsReadyEvent_ = other.tophitsReadyEvent_;
    other.tophitsReadyEvent_ = 0;
    tophitsCopiedEvent_ = other.tophitsCopiedEvent_;
    other.tophitsCopiedEvent_ = 0;
}
//---------------------------------------------------------------
template<class Location>
query_batch<Location>::query_gpu_data::~query_gpu_data()
{
    if(queryIds_)          hipFree    (queryIds_);
    if(sequenceOffsets_)   hipFree    (sequenceOffsets_);
    if(sequences_)         hipFree    (sequences_);
    if(sketches_)          hipFree    (sketches_);
    if(queryResults_)      hipFree    (queryResults_);
    if(queryResultsTmp_)   hipFree    (queryResultsTmp_);
    if(resultOffsets_)     hipFree    (resultOffsets_);
    if(resultCounts_)      hipFree    (resultCounts_);
    if(segBinCounters_)    hipFree    (segBinCounters_);
    if(topCandidates_)     hipFree    (topCandidates_);
    if(maxWindowsInRange_) hipFree    (maxWindowsInRange_);
    CUERR
    if(workStream_) hipStreamDestroy(workStream_);
    if(copyStream_) hipStreamDestroy(copyStream_);
    CUERR
    if(queryFinishedEvent_)  hipEventDestroy(queryFinishedEvent_);
    if(sketchesCopiedEvent_) hipEventDestroy(sketchesCopiedEvent_);
    if(queryIdsCopiedEvent_)  hipEventDestroy(queryIdsCopiedEvent_);
    if(queryIdsFinishedEvent_) hipEventDestroy(queryIdsFinishedEvent_);
    if(offsetsReadyEvent_)  hipEventDestroy(offsetsReadyEvent_);
    if(offsetsCopiedEvent_) hipEventDestroy(offsetsCopiedEvent_);
    if(allhitsReadyEvent_)   hipEventDestroy(allhitsReadyEvent_);
    if(allhitsCopiedEvent_)  hipEventDestroy(allhitsCopiedEvent_);
    if(tophitsReadyEvent_)   hipEventDestroy(tophitsReadyEvent_);
    if(tophitsCopiedEvent_)  hipEventDestroy(tophitsCopiedEvent_);
    CUERR
}


//---------------------------------------------------------------
template<class Location>
query_batch<Location>::query_batch(
    index_type maxQueries,
    size_type maxSequenceLength,
    size_type maxSketchSize,
    size_type maxResultsPerWindow,
    size_type maxCandidatesPerQuery,
    bool copyAllHits,
    part_id numHostThreads,
    part_id numGPUs
) :
    maxWindows_{maxQueries},
    maxSequenceLength_{maxSequenceLength},
    maxSketchSize_{maxSketchSize},
    maxResultsPerWindow_{maxResultsPerWindow},
    maxCandidatesPerQuery_{maxCandidatesPerQuery},
    hostData_{},
    gpuData_{},
    sorters_{},
    numGPUs_{numGPUs}
{
    hipStreamCreate(&h2dCopyStream_);

    hipEventCreate(&queriesCopiedEvent_);
    hipEventCreate(&queryIdsCopiedEvent_);
    hipEventCreate(&maxWinCopiedEvent_);
    CUERR

    hipSetDevice(numGPUs_-1); CUERR

    for(part_id hostId = 0; hostId < numHostThreads; ++hostId) {
        hostData_.emplace_back(maxQueries, maxSequenceLength, maxResultsPerWindow, maxCandidatesPerQuery, copyAllHits);
    }

    using location_type_equivalent = uint64_t;
    static_assert(sizeof(location_type) == sizeof(location_type_equivalent), "location_type must be 64 bit");

    for(part_id gpuId = 0; gpuId < numGPUs_; ++gpuId) {
        hipSetDevice(gpuId); CUERR

        gpuData_.emplace_back(maxQueries, maxSequenceLength, maxSketchSize, maxResultsPerWindow, maxCandidatesPerQuery, numGPUs > 1, gpuId);

        sorters_.emplace_back(
            (location_type_equivalent*)(gpuData_[gpuId].queryResultsTmp_),
            (location_type_equivalent*)(gpuData_[gpuId].queryResults_),
            gpuData_[gpuId].resultOffsets_,
            gpuData_[gpuId].resultCounts_, // reuse for binning
            gpuData_[gpuId].segBinCounters_,
            gpuData_[gpuId].workStream_);
        CUERR
    }
}
//---------------------------------------------------------------
template<class Location>
query_batch<Location>::~query_batch()
{
    hipStreamDestroy(h2dCopyStream_);
    CUERR
    hipEventDestroy(queriesCopiedEvent_);
    hipEventDestroy(queryIdsCopiedEvent_);
    hipEventDestroy(maxWinCopiedEvent_);
    CUERR
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::copy_queries_to_device_async(part_id hostId)
{
    // copy from host to device 0
    part_id gpuId = 0;

    hipSetDevice(gpuId);

    hipStreamWaitEvent(h2dCopyStream_, gpuData_[gpuId].queryFinishedEvent_, 0);

    hipMemcpyAsync(gpuData_[gpuId].sequenceOffsets_, hostData_[hostId].sequence_offsets(),
                    (hostData_[hostId].num_windows()+1)*sizeof(size_type),
                    hipMemcpyHostToDevice, h2dCopyStream_);
    hipMemcpyAsync(gpuData_[gpuId].sequences_, hostData_[hostId].sequences(),
                    hostData_[hostId].sequence_offsets()[hostData_[hostId].num_windows()]*sizeof(char),
                    hipMemcpyHostToDevice, h2dCopyStream_);

    hipEventRecord(queriesCopiedEvent_, h2dCopyStream_);

    hipStreamWaitEvent(gpuData_[gpuId].workStream_, queriesCopiedEvent_, 0);
    hipStreamWaitEvent(gpuData_[gpuId].workStream_, gpuData_[gpuId].sketchesCopiedEvent_, 0);

    // hipEventSynchronize(queriesCopiedEvent_);
    // CUERR


    hipStreamWaitEvent(h2dCopyStream_, gpuData_[gpuId].queryIdsCopiedEvent_, 0);
    hipStreamWaitEvent(h2dCopyStream_, gpuData_[gpuId].queryIdsFinishedEvent_, 0);

    hipMemcpyAsync(gpuData_[gpuId].queryIds_, hostData_[hostId].query_ids(),
                    hostData_[hostId].num_windows()*sizeof(index_type),
                    hipMemcpyHostToDevice, h2dCopyStream_);

    hipEventRecord(queryIdsCopiedEvent_, h2dCopyStream_);

    // hipEventSynchronize(queryIdsCopiedEvent_);
    // CUERR


    hipStreamWaitEvent(h2dCopyStream_, gpuData_[gpuId].tophitsCopiedEvent_, 0);

    hipMemcpyAsync(gpuData_[gpuId].maxWindowsInRange_, hostData_[hostId].max_windows_in_range(),
                    hostData_[hostId].num_queries()*sizeof(window_id),
                    hipMemcpyHostToDevice, h2dCopyStream_);

    hipEventRecord(maxWinCopiedEvent_, h2dCopyStream_);

    // hipEventSynchronize(maxWinCopiedEvent_);
    // CUERR
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::copy_queries_to_next_device_async(part_id hostId, part_id gpuId)
{
    // copy from device gpuId to device gpuId+1
    hipSetDevice(gpuId);

    if(gpuId == 0)
        hipStreamWaitEvent(gpuData_[gpuId].copyStream_, gpuData_[gpuId].queryFinishedEvent_, 0);

    hipStreamWaitEvent(gpuData_[gpuId].copyStream_, gpuData_[gpuId+1].queryFinishedEvent_, 0);
    hipStreamWaitEvent(gpuData_[gpuId].copyStream_, gpuData_[gpuId+1].sketchesCopiedEvent_, 0);


    hipMemcpyPeerAsync(gpuData_[gpuId+1].sketches_, gpuId+1,
                        gpuData_[gpuId].sketches_, gpuId,
                        hostData_[hostId].num_windows()*maxSketchSize_*sizeof(feature_type),
                        gpuData_[gpuId].copyStream_);

    hipEventRecord(gpuData_[gpuId].sketchesCopiedEvent_, gpuData_[gpuId].copyStream_);

    // hipEventSynchronize(gpuData_[gpuId].sketchesCopiedEvent_);
    // CUERR

    hipSetDevice(gpuId+1);

    hipStreamWaitEvent(gpuData_[gpuId+1].copyStream_, gpuData_[gpuId].sketchesCopiedEvent_, 0);
    hipStreamWaitEvent(gpuData_[gpuId+1].workStream_, gpuData_[gpuId].sketchesCopiedEvent_, 0);


    hipSetDevice(gpuId);

    if(gpuId == 0)
        hipStreamWaitEvent(gpuData_[gpuId].copyStream_, queryIdsCopiedEvent_, 0);
    else
        hipStreamWaitEvent(gpuData_[gpuId].copyStream_, gpuData_[gpuId-1].queryIdsCopiedEvent_, 0);

    hipStreamWaitEvent(gpuData_[gpuId].copyStream_, gpuData_[gpuId+1].queryIdsCopiedEvent_, 0);
    hipStreamWaitEvent(gpuData_[gpuId].copyStream_, gpuData_[gpuId+1].queryIdsFinishedEvent_, 0);

    hipMemcpyPeerAsync(gpuData_[gpuId+1].queryIds_, gpuId+1,
                        gpuData_[gpuId].queryIds_, gpuId,
                        hostData_[hostId].num_windows()*sizeof(index_type),
                        gpuData_[gpuId].copyStream_);

    hipEventRecord(gpuData_[gpuId].queryIdsCopiedEvent_, gpuData_[gpuId].copyStream_);

    // hipEventSynchronize(gpuData_[gpuId].queryIdsCopiedEvent_);
    // CUERR
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::wait_for_queries_copied()
{
    // hipEventSynchronize(queriesCopiedEvent_);
    hipEventSynchronize(maxWinCopiedEvent_);
    CUERR
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::mark_query_finished(part_id gpuId)
{
    hipEventRecord(gpuData_[gpuId].queryFinishedEvent_, gpuData_[gpuId].workStream_);
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::compact_results_async(part_id hostId, part_id gpuId)
{
    hipSetDevice(gpuId);

    if(gpuId == 0)
        hipStreamWaitEvent(gpuData_[gpuId].workStream_, queryIdsCopiedEvent_, 0);
    else
        hipStreamWaitEvent(gpuData_[gpuId].workStream_, gpuData_[gpuId-1].queryIdsCopiedEvent_, 0);

    size_t tempStorageBytes = maxWindows_*maxResultsPerWindow_*sizeof(location_type);
    void * d_tempStorage = (void*)(gpuData_[gpuId].queryResultsTmp_);

    hipError_t err = hipcub::DeviceScan::InclusiveSum(
        d_tempStorage, tempStorageBytes,
        gpuData_[gpuId].resultCounts_, gpuData_[gpuId].resultCounts_,
        hostData_[hostId].num_windows(),
        gpuData_[gpuId].workStream_
    );
    // hipStreamSynchronize(gpuData_[gpuId].workStream_);
    // CUERR

    if (err != hipSuccess) {                       \
        std::cout << "CUDA error: " << hipGetErrorString(err) << " : "    \
        << __FILE__ << ", line " << __LINE__ << std::endl;       \
        exit(1);                                                           \
    }

    compact_results<<<hostData_[hostId].num_windows(),128,0,gpuData_[gpuId].workStream_>>>(
        hostData_[hostId].num_windows(),
        gpuData_[gpuId].resultCounts_,
        maxResultsPerWindow_,
        gpuData_[gpuId].queryResults_,
        gpuData_[gpuId].queryResultsTmp_,
        gpuData_[gpuId].queryIds_,
        gpuData_[gpuId].resultOffsets_);

    hipEventRecord(gpuData_[gpuId].queryIdsFinishedEvent_, gpuData_[gpuId].workStream_);

    // hipEventSynchronize(gpuData_[gpuId].queryIdsFinishedEvent_);
    // CUERR
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::compact_sort_and_copy_allhits_async(
    part_id hostId,
    part_id gpuId,
    bool copyAllHits)
{
    hipSetDevice(gpuId);

    compact_results_async(hostId, gpuId);
    // hipStreamSynchronize(gpuData_[gpuId].workStream_);
    // CUERR

    if(copyAllHits) {
        hipEventRecord(gpuData_[gpuId].offsetsReadyEvent_, gpuData_[gpuId].workStream_);
        hipStreamWaitEvent(gpuData_[gpuId].copyStream_, gpuData_[gpuId].offsetsReadyEvent_, 0);

        hipMemcpyAsync(hostData_[hostId].result_offsets(), gpuData_[gpuId].resultOffsets_,
                        (hostData_[hostId].num_queries()+1)*sizeof(int),
                        hipMemcpyDeviceToHost, gpuData_[gpuId].copyStream_);

        hipEventRecord(gpuData_[gpuId].offsetsCopiedEvent_, gpuData_[gpuId].copyStream_);

        // hipStreamSynchronize(gpuData_[gpuId].copyStream_);
        // CUERR
    }

    sorters_[gpuId].run(hostData_[hostId].num_queries(), hostData_[hostId].largest_segment_size(), gpuData_[gpuId].workStream_);
    // hipStreamSynchronize(gpuData_[gpuId].workStream_);
    // CUERR

    if(copyAllHits) {
        hipEventRecord(gpuData_[gpuId].allhitsReadyEvent_, gpuData_[gpuId].workStream_);
        hipStreamWaitEvent(gpuData_[gpuId].copyStream_, gpuData_[gpuId].allhitsReadyEvent_, 0);

        hipEventSynchronize(gpuData_[gpuId].offsetsCopiedEvent_);

        hipMemcpyAsync(hostData_[hostId].query_results(), gpuData_[gpuId].queryResults_,
                        hostData_[hostId].result_offsets()[hostData_[hostId].num_queries()]*sizeof(location_type),
                        hipMemcpyDeviceToHost, gpuData_[gpuId].copyStream_);

        hipEventRecord(gpuData_[gpuId].allhitsCopiedEvent_, gpuData_[gpuId].copyStream_);

        // hipStreamSynchronize(gpuData_[gpuId].copyStream_);
        // CUERR
    }
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::generate_and_copy_top_candidates_async(
    part_id hostId,
    part_id gpuId,
    const ranked_lineage * lineages,
    taxon_rank lowestRank)
{
    hipSetDevice(gpuId);

    hipEvent_t& event = (gpuId == 0) ? maxWinCopiedEvent_ : gpuData_[gpuId-1].tophitsCopiedEvent_;

    hipStreamWaitEvent(gpuData_[gpuId].workStream_, event, 0);
    if(gpuId+1 < numGPUs_) {
        // copy maxWindowsInRange to next device
        hipStreamWaitEvent(gpuData_[gpuId].copyStream_, event, 0);
        hipStreamWaitEvent(gpuData_[gpuId].copyStream_, gpuData_[gpuId+1].tophitsCopiedEvent_, 0);

        hipMemcpyPeerAsync(gpuData_[gpuId+1].maxWindowsInRange_, gpuId+1,
            gpuData_[gpuId].maxWindowsInRange_, gpuId,
            hostData_[hostId].num_queries()*sizeof(window_id),
            gpuData_[gpuId].copyStream_);
    }

    const index_type numBlocks = hostData_[hostId].num_queries();

    //TODO different max cand cases
    if(maxCandidatesPerQuery_ <= 2) {
        constexpr int maxCandidates = 2;

        bool update = gpuId > 0;

        generate_top_candidates<maxCandidates><<<numBlocks,32,0,gpuData_[gpuId].workStream_>>>(
            hostData_[hostId].num_queries(),
            gpuData_[gpuId].resultOffsets_,
            gpuData_[gpuId].queryResults_,
            gpuData_[gpuId].maxWindowsInRange_,
            lineages,
            lowestRank,
            maxCandidatesPerQuery_,
            gpuData_[gpuId].topCandidates_,
            update);

        // hipStreamSynchronize(gpuData_[gpuId].workStream_);
        // CUERR
    }
    else {
        std::cerr << "At most 2 candidates per query allowed!\n";
    }

    hipEventRecord(gpuData_[gpuId].tophitsReadyEvent_, gpuData_[gpuId].workStream_);
    hipStreamWaitEvent(gpuData_[gpuId].copyStream_, gpuData_[gpuId].tophitsReadyEvent_, 0);

    if(gpuId == numGPUs_-1) {
        // copy candidates from last device to host
        hipMemcpyAsync(hostData_[hostId].top_candidates(), gpuData_[gpuId].topCandidates_,
                        hostData_[hostId].num_queries()*maxCandidatesPerQuery_*sizeof(match_candidate),
                        hipMemcpyDeviceToHost, gpuData_[gpuId].copyStream_);

        hipEventRecord(hostData_[hostId].results_copied_event(), gpuData_[gpuId].copyStream_);
    }
    else {
        // copy candidates to next device
        hipMemcpyPeerAsync(gpuData_[gpuId+1].topCandidates_, gpuId+1,
                            gpuData_[gpuId].topCandidates_, gpuId,
                            hostData_[hostId].num_queries()*maxCandidatesPerQuery_*sizeof(match_candidate),
                            gpuData_[gpuId].copyStream_);
    }

    hipEventRecord(gpuData_[gpuId].tophitsCopiedEvent_, gpuData_[gpuId].copyStream_);

    // hipStreamSynchronize(gpuData_[gpuId].copyStream_);
    // CUERR
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::wait_for_allhits_copied(part_id gpuId)
{
    hipStreamWaitEvent(gpuData_[gpuId].workStream_, gpuData_[gpuId].allhitsCopiedEvent_, 0);
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::sync_work_stream(part_id gpuId)
{
    hipStreamSynchronize(gpuData_[gpuId].workStream_);
}

//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::sync_copy_stream(part_id gpuId)
{
    hipStreamSynchronize(gpuData_[gpuId].copyStream_);
}


//---------------------------------------------------------------
template class query_batch<location>;

} // namespace mc
