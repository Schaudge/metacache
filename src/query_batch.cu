#include "hip/hip_runtime.h"

#include "query_batch.cuh"
#include "sketch_database.h"
#include "gpu_result_processing.cuh"

#include "cub/device/device_scan.cuh"

#include "../dep/bb_segsort/src/bb_segsort_keys.cuh"

namespace mc {


//---------------------------------------------------------------
template<class Location>
class query_batch<Location>::segmented_sort
{
    using location_type_equivalent = uint64_t;

    static_assert(sizeof(location_type) == sizeof(location_type_equivalent), "location_type must be 64 bit");

public:
    segmented_sort(
        location_type_equivalent *d_keys, location_type_equivalent *d_keysB,
        const int *d_segs,
        int *d_binnedSegIds,
        int *d_segBinCounters_,
        hipStream_t stream)
    :
        sorter_{d_keys, d_keysB,
            d_segs, d_binnedSegIds, d_segBinCounters_,
            stream}
    {}

    void run(int numSegs, hipStream_t stream) const {
        sorter_.run(numSegs, stream);
    }

private:
    bb_segsort_keys<location_type_equivalent> sorter_;
};



//---------------------------------------------------------------
template<class Location>
query_batch<Location>::query_batch(
    index_type maxQueries,
    size_type maxSequenceLength,
    size_type maxResultsPerQuery,
    size_type maxCandidatesPerQuery
) :
    hostInput_{},
    hostOutput_{},
    gpuData_{},
    maxQueries_{maxQueries},
    maxSequenceLength_{maxSequenceLength},
    maxResultsPerQuery_{maxResultsPerQuery},
    maxCandidatesPerQuery_{maxCandidatesPerQuery}
{
    hostInput_.numSegments_ = 0;
    hostOutput_.numSegments_ = 0;
    gpuData_.numSegments_ = 0;

    hostInput_.numQueries_ = 0;
    gpuData_.numQueries_ = 0;

    size_t allocatedGpuMem = 0;

    if(maxQueries_ && maxSequenceLength_ && maxResultsPerQuery_) {
        hipHostMalloc(&hostInput_.queryIds_, maxQueries_*sizeof(index_type));
        hipMalloc    (&gpuData_.queryIds_, maxQueries_*sizeof(index_type));
        allocatedGpuMem += maxQueries_*sizeof(index_type);

        hipHostMalloc(&hostInput_.sequenceOffsets_, (maxQueries_+1)*sizeof(size_type));
        hipMalloc    (&gpuData_.sequenceOffsets_, (maxQueries_+1)*sizeof(size_type));
        allocatedGpuMem += (maxQueries_+1)*sizeof(size_type);
        hostInput_.sequenceOffsets_[0] = 0;

        hipHostMalloc(&hostInput_.sequences_, maxSequenceLength_*sizeof(char));
        hipMalloc    (&gpuData_.sequences_, maxSequenceLength_*sizeof(char));
        allocatedGpuMem += maxSequenceLength_*sizeof(char);

        hipHostMalloc(&hostOutput_.queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(location_type));
        hipMalloc    (&gpuData_.queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(location_type));
        hipMalloc    (&gpuData_.queryResultsTmp_, maxQueries_*maxResultsPerQuery_*sizeof(location_type));
        allocatedGpuMem += 2*maxQueries_*maxResultsPerQuery_*sizeof(location_type);

        hipHostMalloc(&hostOutput_.resultOffsets_, (maxQueries_+1)*sizeof(int));
        hipMalloc    (&gpuData_.resultOffsets_, (maxQueries_+1)*sizeof(int));
        allocatedGpuMem += (maxQueries_+1)*sizeof(int);
        hostOutput_.resultOffsets_[0] = 0;
        hipMemcpy(gpuData_.resultOffsets_, hostOutput_.resultOffsets_, sizeof(int), hipMemcpyHostToDevice);

        hipMalloc    (&gpuData_.resultCounts_, maxQueries_*sizeof(int));
        allocatedGpuMem += maxQueries_*sizeof(int);

        hipMalloc    (&gpuData_.segBinCounters_, (SEGBIN_NUM+1)*sizeof(int));
        allocatedGpuMem += (SEGBIN_NUM+1)*sizeof(int);

        hipHostMalloc(&hostOutput_.topCandidates_, maxQueries_*maxCandidatesPerQuery_*sizeof(match_candidate));
        hipMalloc    (&gpuData_.topCandidates_, maxQueries_*maxCandidatesPerQuery_*sizeof(match_candidate));
        allocatedGpuMem += maxQueries_*maxCandidatesPerQuery_*sizeof(match_candidate);

        hipHostMalloc(&hostInput_.maxWindowsInRange_, maxQueries_*sizeof(window_id));
        hipMalloc    (&gpuData_.maxWindowsInRange_, maxQueries_*sizeof(window_id));
        allocatedGpuMem += maxQueries_*sizeof(window_id);
    }
    CUERR

    // std::cerr << "query batch size on gpu: " << (allocatedGpuMem >> 20) << " MB\n";

    hipStreamCreate(&stream_);
    hipStreamCreate(&resultCopyStream_);

    hipEventCreate(&queriesCopiedEvent_);
    hipEventCreate(&offsetsCopiedEvent_);
    hipEventCreate(&resultReadyEvent_);
    CUERR

    using location_type_equivalent = uint64_t;

    static_assert(sizeof(location_type) == sizeof(location_type_equivalent), "location_type must be 64 bit");

    sorter_ = std::make_unique<segmented_sort>(
        (location_type_equivalent*)(gpuData_.queryResultsTmp_),
        (location_type_equivalent*)(gpuData_.queryResults_),
        gpuData_.resultOffsets_,
        gpuData_.resultCounts_, // reuse for binning
        gpuData_.segBinCounters_,
        stream_);
    CUERR
}
//---------------------------------------------------------------
template<class Location>
query_batch<Location>::~query_batch() {
    CUERR

    if(maxQueries_ && maxSequenceLength_ && maxResultsPerQuery_) {
        hipHostFree(hostInput_.queryIds_);
        hipFree    (gpuData_.queryIds_);

        hipHostFree(hostInput_.sequenceOffsets_);
        hipFree    (gpuData_.sequenceOffsets_);

        hipHostFree(hostInput_.sequences_);
        hipFree    (gpuData_.sequences_);

        hipHostFree(hostOutput_.queryResults_);
        hipFree    (gpuData_.queryResults_);

        hipFree    (gpuData_.queryResultsTmp_);

        hipHostFree(hostOutput_.resultOffsets_);
        hipFree    (gpuData_.resultOffsets_);

        hipFree    (gpuData_.resultCounts_);

        hipFree    (gpuData_.segBinCounters_);

        hipHostFree(hostOutput_.topCandidates_);
        hipFree    (gpuData_.topCandidates_);

        hipHostFree(hostInput_.maxWindowsInRange_);
        hipFree    (gpuData_.maxWindowsInRange_);
    }
    CUERR

    hipStreamDestroy(stream_);
    hipStreamDestroy(resultCopyStream_);

    hipEventDestroy(queriesCopiedEvent_);
    hipEventDestroy(offsetsCopiedEvent_);
    hipEventDestroy(resultReadyEvent_);
    CUERR
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::copy_queries_to_device_async() {
    gpuData_.numQueries_ = hostInput_.numQueries_;
    gpuData_.numSegments_ = hostInput_.numSegments_;
    hostOutput_.numSegments_ = hostInput_.numSegments_;

    hipMemcpyAsync(gpuData_.queryIds_, hostInput_.queryIds_,
                    gpuData_.numQueries_*sizeof(index_type),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(gpuData_.sequenceOffsets_, hostInput_.sequenceOffsets_,
                    (gpuData_.numQueries_+1)*sizeof(size_type),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(gpuData_.sequences_, hostInput_.sequences_,
                    hostInput_.sequenceOffsets_[gpuData_.numQueries_]*sizeof(char),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(gpuData_.maxWindowsInRange_, hostInput_.maxWindowsInRange_,
                    gpuData_.numSegments_*sizeof(window_id),
                    hipMemcpyHostToDevice, stream_);

    hipEventRecord(queriesCopiedEvent_, stream_);

    // hipStreamSynchronize(stream_);
    // CUERR
}

//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::wait_for_queries_copied() {
    hipEventSynchronize(queriesCopiedEvent_);
}



//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::sync_streams() {
    hipStreamSynchronize(stream_);
    hipStreamSynchronize(resultCopyStream_);
}

//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::sync_result_stream() {
    hipStreamSynchronize(resultCopyStream_);
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::compact_results_async() {

    size_t tempStorageBytes = maxQueries_*maxResultsPerQuery_*sizeof(location_type);
    void * d_tempStorage = (void*)(gpuData_.queryResultsTmp_);

    hipError_t err = hipcub::DeviceScan::InclusiveSum(
        d_tempStorage, tempStorageBytes,
        gpuData_.resultCounts_, gpuData_.resultCounts_,
        gpuData_.numQueries_,
        stream_
    );
    // hipStreamSynchronize(stream_);
    // CUERR

    if (err != hipSuccess) {                       \
        std::cout << "CUDA error: " << hipGetErrorString(err) << " : "    \
        << __FILE__ << ", line " << __LINE__ << std::endl;       \
        exit(1);                                                           \
    }

    compact_kernel<<<gpuData_.numQueries_,128,0,stream_>>>(
        gpuData_.numQueries_,
        gpuData_.resultCounts_,
        maxResultsPerQuery_,
        gpuData_.queryResults_,
        gpuData_.queryResultsTmp_,
        gpuData_.queryIds_,
        gpuData_.resultOffsets_);
    // hipStreamSynchronize(stream_);
    // CUERR
}



//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::compact_sort_and_copy_allhits_async(bool copyAllHits)
{
    compact_results_async();

    if(copyAllHits) {
        hipEventRecord(resultReadyEvent_, stream_);
        hipStreamWaitEvent(resultCopyStream_, resultReadyEvent_, 0);

        hipMemcpyAsync(hostOutput_.resultOffsets_, gpuData_.resultOffsets_,
                        (gpuData_.numSegments_+1)*sizeof(int),
                        hipMemcpyDeviceToHost, resultCopyStream_);
    }
    // hipStreamSynchronize(resultCopyStream_);
    // CUERR

    sorter_->run(gpuData_.numSegments_, stream_);
    // hipStreamSynchronize(stream_);
    // CUERR

    if(copyAllHits) {
        hipEventRecord(offsetsCopiedEvent_, resultCopyStream_);

        hipEventRecord(resultReadyEvent_, stream_);
        hipStreamWaitEvent(resultCopyStream_, resultReadyEvent_, 0);

        hipEventSynchronize(offsetsCopiedEvent_);

        hipMemcpyAsync(hostOutput_.queryResults_, gpuData_.queryResults_,
                        hostOutput_.resultOffsets_[gpuData_.numSegments_]*sizeof(location_type),
                        hipMemcpyDeviceToHost, resultCopyStream_);
    }
    // hipStreamSynchronize(resultCopyStream_);
    // CUERR
}


//---------------------------------------------------------------
template<class Location>
void query_batch<Location>::generate_and_copy_top_candidates_async(
    const ranked_lineage * lineages,
    taxon_rank lowestRank)
{
    const index_type numBlocks = gpuData_.numSegments_;

    //TODO different max cand cases
    if(maxCandidatesPerQuery_ <= 2) {
        constexpr int maxCandidates = 2;

        generate_top_candidates<maxCandidates><<<numBlocks,32,0,stream_>>>(
            gpuData_.numSegments_,
            gpuData_.resultOffsets_,
            gpuData_.queryResults_,
            gpuData_.maxWindowsInRange_,
            lineages,
            lowestRank,
            maxCandidatesPerQuery_,
            gpuData_.topCandidates_);

        // hipStreamSynchronize(stream_);
        // CUERR
    }

    hipEventRecord(resultReadyEvent_, stream_);
    hipStreamWaitEvent(resultCopyStream_, resultReadyEvent_, 0);

    // copy candidates to host
    hipMemcpyAsync(hostOutput_.topCandidates_, gpuData_.topCandidates_,
                    gpuData_.numSegments_*maxCandidatesPerQuery_*sizeof(match_candidate),
                    hipMemcpyDeviceToHost, resultCopyStream_);

    // hipStreamSynchronize(resultCopyStream_);
    // CUERR
}


//---------------------------------------------------------------
template class query_batch<location>;

} // namespace mc
