#include "hip/hip_runtime.h"

#include "query_batch.cuh"
#include "sketch_database.h"
#include "gpu_result_processing.cuh"

#include "../dep/cub/cub/device/device_scan.cuh"

#include "../dep/bb_segsort/src/bb_segsort_keys.cuh"

namespace mc {


//---------------------------------------------------------------
template<class result_type>
class query_batch<result_type>::segmented_sort
{
    using result_type_equivalent = uint64_t;

    static_assert(sizeof(result_type) == sizeof(result_type_equivalent), "result_type must be 64 bit");

public:
    segmented_sort(
        result_type_equivalent *d_keys, result_type_equivalent *d_keysB,
        const int *d_segs,
        int *d_bin_segs_id, int *d_bin_counter,
        hipStream_t stream)
    :
        sorter_(d_keys, d_keysB,
            d_segs, d_bin_segs_id, d_bin_counter,
            stream)
    {}

    void run(int num_segs, hipStream_t stream) const {
        sorter_.run(num_segs, stream);
    }

private:
    bb_segsort_keys<result_type_equivalent> sorter_;
};



//---------------------------------------------------------------
template<class result_type>
query_batch<result_type>::query_batch(
    id_type maxQueries,
    size_t maxSequenceLength,
    size_t maxResultsPerQuery,
    uint32_t maxCandidatesPerQuery
) :
    h_numSegments_{0},
    d_numSegments_{0},
    h_numQueries_{0},
    d_numQueries_{0},
    maxQueries_{maxQueries},
    maxSequenceLength_{maxSequenceLength},
    maxResultsPerQuery_{maxResultsPerQuery},
    maxCandidatesPerQuery_{maxCandidatesPerQuery}
{
    //TODO reuse/combine device arrays:
    // d_sequenceOffsets_ + d_resultOffsets_

    if(maxQueries_ && maxSequenceLength_ && maxResultsPerQuery_) {
        hipHostMalloc(&h_queryIds_, maxQueries_*sizeof(id_type));
        hipMalloc    (&d_queryIds_, maxQueries_*sizeof(id_type));

        hipHostMalloc(&h_sequenceOffsets_, (maxQueries_+1)*sizeof(encodinglen_t));
        hipMalloc    (&d_sequenceOffsets_, (maxQueries_+1)*sizeof(encodinglen_t));
        h_sequenceOffsets_[0] = 0;

        hipHostMalloc(&h_sequences_, maxSequenceLength_*sizeof(char));
        hipMalloc    (&d_sequences_, maxSequenceLength_*sizeof(char));

        hipHostMalloc(&h_queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));
        hipMalloc    (&d_queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));

        hipMalloc    (&d_queryResultsTmp_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));

        hipHostMalloc(&h_resultOffsets_, (maxQueries_+1)*sizeof(int));
        hipMalloc    (&d_resultOffsets_, (maxQueries_+1)*sizeof(int));
        h_resultOffsets_[0] = 0;
        hipMemcpy(d_resultOffsets_, h_resultOffsets_, sizeof(int), hipMemcpyHostToDevice);

        hipMalloc    (&d_resultCounts_, maxQueries_*sizeof(int));
        d_binnedSegIds_ = d_resultCounts_;

        hipHostMalloc(&h_segBinCounters_, (SEGBIN_NUM+1)*sizeof(int));
        hipMalloc    (&d_segBinCounters_, (SEGBIN_NUM+1)*sizeof(int));

        hipHostMalloc(&h_topCandidates_, maxQueries_*maxCandidatesPerQuery_*sizeof(match_candidate));
        hipMalloc    (&d_topCandidates_, maxQueries_*maxCandidatesPerQuery_*sizeof(match_candidate));

        hipHostMalloc(&h_maxWindowsInRange_, maxQueries_*sizeof(window_id));
        hipMalloc    (&d_maxWindowsInRange_, maxQueries_*sizeof(window_id));
    }
    CUERR

    hipStreamCreate(&stream_);
    hipStreamCreate(&resultCopyStream_);

    hipEventCreate(&queriesCopiedEvent_);
    hipEventCreate(&offsetsCopiedEvent_);
    hipEventCreate(&resultReadyEvent_);
    CUERR

    using result_type_equivalent = uint64_t;

    static_assert(sizeof(result_type) == sizeof(result_type_equivalent), "result_type must be 64 bit");

    sorter_ = std::make_unique<segmented_sort>(
        (result_type_equivalent*)(d_queryResultsTmp_),
        (result_type_equivalent*)(d_queryResults_),
        d_resultOffsets_,
        d_binnedSegIds_, d_segBinCounters_,
        stream_);
}
//---------------------------------------------------------------
template<class result_type>
query_batch<result_type>::~query_batch() {
    CUERR

    if(maxQueries_ && maxSequenceLength_ && maxResultsPerQuery_) {
        hipHostFree(h_queryIds_);
        hipFree    (d_queryIds_);

        hipHostFree(h_sequenceOffsets_);
        hipFree    (d_sequenceOffsets_);

        hipHostFree(h_sequences_);
        hipFree    (d_sequences_);

        hipHostFree(h_queryResults_);
        hipFree    (d_queryResults_);

        hipFree    (d_queryResultsTmp_);

        hipHostFree(h_resultOffsets_);
        hipFree    (d_resultOffsets_);

        hipFree    (d_resultCounts_);
        d_binnedSegIds_ = nullptr;

        hipHostFree(h_segBinCounters_);
        hipFree    (d_segBinCounters_);

        hipHostFree(h_topCandidates_);
        hipFree    (d_topCandidates_);

        hipHostFree(h_maxWindowsInRange_);
        hipFree    (d_maxWindowsInRange_);
    }
    CUERR

    hipStreamDestroy(stream_);
    hipStreamDestroy(resultCopyStream_);

    hipEventDestroy(queriesCopiedEvent_);
    hipEventDestroy(offsetsCopiedEvent_);
    hipEventDestroy(resultReadyEvent_);
    CUERR
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::copy_queries_to_device_async() {
    d_numQueries_ = h_numQueries_;
    d_numSegments_ = h_numSegments_;

    hipMemcpyAsync(d_queryIds_, h_queryIds_,
                    d_numQueries_*sizeof(id_type),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_sequenceOffsets_, h_sequenceOffsets_,
                    (d_numQueries_+1)*sizeof(encodinglen_t),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_sequences_, h_sequences_,
                    h_sequenceOffsets_[d_numQueries_]*sizeof(char),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_maxWindowsInRange_, h_maxWindowsInRange_,
                    d_numSegments_*sizeof(window_id),
                    hipMemcpyHostToDevice, stream_);

    hipEventRecord(queriesCopiedEvent_, stream_);

    // hipStreamSynchronize(stream_);
    // CUERR
}

//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::wait_for_queries_copied() {
    hipEventSynchronize(queriesCopiedEvent_);
}



//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::sync_streams() {
    hipStreamSynchronize(stream_);
    hipStreamSynchronize(resultCopyStream_);
}

//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::sync_result_stream() {
    hipStreamSynchronize(resultCopyStream_);
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::compact_results_async() {

    size_t tempStorageBytes = maxQueries_*maxResultsPerQuery_*sizeof(result_type);
    void * d_tempStorage = (void*)(d_queryResultsTmp_);

    hipError_t err = hipcub::DeviceScan::InclusiveSum(
        d_tempStorage, tempStorageBytes,
        d_resultCounts_, d_resultCounts_,
        d_numQueries_,
        stream_
    );
    // hipStreamSynchronize(stream_);
    // CUERR

    if (err != hipSuccess) {                       \
        std::cout << "CUDA error: " << hipGetErrorString(err) << " : "    \
        << __FILE__ << ", line " << __LINE__ << std::endl;       \
        exit(1);                                                           \
    }

    compact_kernel<<<d_numQueries_,32,0,stream_>>>(
        d_numQueries_,
        d_resultCounts_,
        maxResultsPerQuery_,
        d_queryResults_,
        d_queryResultsTmp_,
        d_queryIds_,
        d_resultOffsets_);
    // hipStreamSynchronize(stream_);
    // CUERR
}



//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::compact_sort_and_copy_results_async(bool copyAllHits)
{
    compact_results_async();

    if(copyAllHits) {
        hipEventRecord(resultReadyEvent_, stream_);
        hipStreamWaitEvent(resultCopyStream_, resultReadyEvent_, 0);

        hipMemcpyAsync(h_resultOffsets_, d_resultOffsets_,
                        (d_numSegments_+1)*sizeof(int),
                        hipMemcpyDeviceToHost, resultCopyStream_);
    }
    // hipStreamSynchronize(resultCopyStream_);
    // CUERR

    sorter_->run(d_numSegments_, stream_);
    // hipStreamSynchronize(stream_);
    // CUERR

    if(copyAllHits) {
        hipEventRecord(offsetsCopiedEvent_, resultCopyStream_);

        hipEventRecord(resultReadyEvent_, stream_);
        hipStreamWaitEvent(resultCopyStream_, resultReadyEvent_, 0);

        hipEventSynchronize(offsetsCopiedEvent_);

        hipMemcpyAsync(h_queryResults_, d_queryResults_,
                        h_resultOffsets_[d_numSegments_]*sizeof(result_type),
                        hipMemcpyDeviceToHost, resultCopyStream_);
    }
    // hipStreamSynchronize(resultCopyStream_);
    // CUERR
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::generate_and_copy_top_candidates_async(
    const ranked_lineage * lineages,
    taxon_rank lowestRank)
{
    const size_t numBlocks = d_numSegments_;

    //TODO different max cand cases
    if(maxCandidatesPerQuery_ <= 2) {
        constexpr int maxCandidates = 2;

        generate_top_candidates<maxCandidates><<<numBlocks,32,0,stream_>>>(
            d_numSegments_,
            d_resultOffsets_,
            d_queryResults_,
            d_maxWindowsInRange_,
            lineages,
            lowestRank,
            maxCandidatesPerQuery_,
            d_topCandidates_);

        // hipStreamSynchronize(stream_);
        // CUERR
    }

    hipEventRecord(resultReadyEvent_, stream_);
    hipStreamWaitEvent(resultCopyStream_, resultReadyEvent_, 0);

    // copy candidates to host
    hipMemcpyAsync(h_topCandidates_, d_topCandidates_,
                    d_numSegments_*maxCandidatesPerQuery_*sizeof(match_candidate),
                    hipMemcpyDeviceToHost, resultCopyStream_);

    // hipStreamSynchronize(resultCopyStream_);
    // CUERR
}


//---------------------------------------------------------------
template class query_batch<location>;

} // namespace mc
