
#include "query_batch.cuh"
#include "gpu_engine.cuh"

namespace mc {


//---------------------------------------------------------------
template<class result_type>
query_batch<result_type>::query_batch(size_t maxQueries, size_t maxEncodeLength, size_t maxResultsPerQuery) :
    numQueries_{0},
    maxQueries_{maxQueries},
    maxEncodeLength_{maxEncodeLength},
    maxResultsPerQuery_{maxResultsPerQuery}
{
    if(maxQueries_) {
        hipHostMalloc(&queryIds_, maxQueries_*sizeof(query_id));

        hipHostMalloc(&h_encodeOffsets_, (maxQueries_+1)*sizeof(encodinglen_t));
        h_encodeOffsets_[0] = 0;
        hipHostMalloc(&h_queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));

        hipMalloc(&d_encodeOffsets_, (maxQueries_+1)*sizeof(encodinglen_t));
        hipMalloc(&d_queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));
    }
    if(maxEncodeLength_) {
        hipHostMalloc(&h_encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));
        hipHostMalloc(&h_encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));

        hipMalloc(&d_encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));
        hipMalloc(&d_encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));
    }
    CUERR
}
//---------------------------------------------------------------
template<class result_type>
query_batch<result_type>::~query_batch() {
    if(maxQueries_) {
        hipHostFree(queryIds_);

        hipHostFree(h_encodeOffsets_);
        hipHostFree(h_queryResults_);

        hipFree(d_encodeOffsets_);
        hipFree(d_queryResults_);
    }
    if(maxEncodeLength_) {
        hipHostFree(h_encodedSeq_);
        hipHostFree(h_encodedAmbig_);

        hipFree(d_encodedSeq_);
        hipFree(d_encodedAmbig_);
    }
    CUERR
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::copy_queries_to_device(hipStream_t stream) {
    hipMemcpyAsync(d_encodeOffsets_, h_encodeOffsets_,
               (numQueries_+1)*sizeof(encodinglen_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_encodedSeq_, h_encodedSeq_,
               h_encodeOffsets_[numQueries_]*sizeof(encodedseq_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_encodedAmbig_, h_encodedAmbig_,
               h_encodeOffsets_[numQueries_]*sizeof(encodedambig_t), hipMemcpyHostToDevice, stream);
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::copy_results_to_host(hipStream_t stream) {
    hipMemcpyAsync(h_queryResults_, d_queryResults_,
               numQueries_*maxResultsPerQuery_*sizeof(result_type), hipMemcpyHostToDevice, stream);
}



//---------------------------------------------------------------
template class query_batch<location>;

} // namespace mc
