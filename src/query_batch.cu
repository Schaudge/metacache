#include "hip/hip_runtime.h"

#include "query_batch.cuh"
#include "sketch_database.h"
#include "gpu_engine.cuh"

#include "../dep/cub/cub/device/device_segmented_radix_sort.cuh"
#include "../dep/cub/cub/device/device_scan.cuh"

namespace mc {


//---------------------------------------------------------------
template<class result_type>
query_batch<result_type>::query_batch(
    id_type maxQueries,
    size_t maxEncodeLength,
    size_t maxResultsPerQuery) :
    numSegments_{0},
    numQueries_{0},
    maxQueries_{maxQueries},
    maxEncodeLength_{maxEncodeLength},
    maxResultsPerQuery_{maxResultsPerQuery}
{
    //TODO reuse/combine device arrays:
    // d_encodeOffsets_ + d_resultOffsets_

    if(maxQueries_ && maxEncodeLength_ && maxResultsPerQuery_) {
        hipHostMalloc(&h_queryIds_, maxQueries_*sizeof(id_type));
        hipMalloc    (&d_queryIds_, maxQueries_*sizeof(id_type));

        hipHostMalloc(&h_encodeOffsets_, (maxQueries_+1)*sizeof(encodinglen_t));
        hipMalloc    (&d_encodeOffsets_, (maxQueries_+1)*sizeof(encodinglen_t));
        h_encodeOffsets_[0] = 0;

        hipHostMalloc(&h_encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));
        hipMalloc    (&d_encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));

        hipHostMalloc(&h_encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));
        hipMalloc    (&d_encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));

        hipHostMalloc(&h_queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));
        hipMalloc    (&d_queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));

        hipMalloc    (&d_queryResultsTmp_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));

        hipHostMalloc(&h_resultOffsets_, (maxQueries_+1)*sizeof(int));
        hipMalloc    (&d_resultOffsets_, (maxQueries_+1)*sizeof(int));
        h_resultOffsets_[0] = 0;
        hipMemcpy(d_resultOffsets_, h_resultOffsets_, sizeof(int), hipMemcpyHostToDevice);

        hipMalloc    (&d_resultCounts_, maxQueries_*sizeof(int));

        hipHostMalloc(&h_topCandidates_, maxQueries_*sizeof(candidate_target));
        hipMalloc    (&d_topCandidates_, maxQueries_*sizeof(candidate_target));

        hipHostMalloc(&h_maxWindowsInRange_, maxQueries_*sizeof(window_id));
        hipMalloc    (&d_maxWindowsInRange_, maxQueries_*sizeof(window_id));
    }
    CUERR

    hipStreamCreate(&stream_);
    hipEventCreate(&event_);
    CUERR
}
//---------------------------------------------------------------
template<class result_type>
query_batch<result_type>::~query_batch() {
    CUERR

    if(maxQueries_ && maxEncodeLength_ && maxResultsPerQuery_) {
        hipHostFree(h_queryIds_);
        hipFree    (d_queryIds_);

        hipHostFree(h_encodeOffsets_);
        hipFree    (d_encodeOffsets_);

        hipHostFree(h_encodedSeq_);
        hipFree    (d_encodedSeq_);

        hipHostFree(h_encodedAmbig_);
        hipFree    (d_encodedAmbig_);

        hipHostFree(h_queryResults_);
        hipFree    (d_queryResults_);

        hipFree    (d_queryResultsTmp_);

        hipHostFree(h_resultOffsets_);
        hipFree    (d_resultOffsets_);

        hipFree    (d_resultCounts_);

        hipHostFree(h_topCandidates_);
        hipFree    (d_topCandidates_);

        hipHostFree(h_maxWindowsInRange_);
        hipFree    (d_maxWindowsInRange_);
    }
    CUERR

    hipStreamDestroy(stream_);
    hipEventDestroy(event_);
    CUERR
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::copy_queries_to_device_async() {
    hipMemcpyAsync(d_queryIds_, h_queryIds_,
                    numQueries_*sizeof(id_type),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_encodeOffsets_, h_encodeOffsets_,
                    (numQueries_+1)*sizeof(encodinglen_t),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_encodedSeq_, h_encodedSeq_,
                    h_encodeOffsets_[numQueries_]*sizeof(encodedseq_t),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_encodedAmbig_, h_encodedAmbig_,
                    h_encodeOffsets_[numQueries_]*sizeof(encodedambig_t),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_maxWindowsInRange_, h_maxWindowsInRange_,
                    numSegments_*sizeof(window_id),
                    hipMemcpyHostToDevice, stream_);
    // hipStreamSynchronize(stream_);
    // CUERR
}



//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::sync_stream() {
    hipStreamSynchronize(stream_);
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::compact_sort_and_copy_results_async() {

    {
        size_t tempStorageBytes = maxQueries_*maxResultsPerQuery_*sizeof(result_type);
        void * d_tempStorage = (void*)(d_queryResultsTmp_);

        // std::cout << "temp size: " << tempStorageBytes << std::endl;

        int numItems = numQueries_;
        // size_t tempStorageBytes = 767;
        hipError_t err = hipcub::DeviceScan::InclusiveSum(
            d_tempStorage, tempStorageBytes,
            d_resultCounts_, d_resultCounts_,
            numItems,
            stream_
        );

        // hipStreamSynchronize(stream_);
        // CUERR

        if (err != hipSuccess) {                       \
            std::cout << "CUDA error: " << hipGetErrorString(err) << " : "    \
                    << __FILE__ << ", line " << __LINE__ << std::endl;       \
            exit(1);                                                           \
        }

        compact_kernel<<<numQueries_,32,0,stream_>>>(
            numQueries_,
            d_resultCounts_,
            maxResultsPerQuery_,
            d_queryResults_,
            d_queryResultsTmp_,
            d_queryIds_,
            d_resultOffsets_
        );

        hipMemcpyAsync(h_resultOffsets_, d_resultOffsets_,
                        (numSegments_+1)*sizeof(int),
                        hipMemcpyDeviceToHost, stream_);

        hipEventRecord(event_, stream_);

        // hipStreamSynchronize(stream_);
        // CUERR
    }

    using result_type_equivalent = uint64_t;

    static_assert(sizeof(result_type) == sizeof(result_type_equivalent), "result_type must be 64 bit");

    hipcub::DoubleBuffer<result_type_equivalent> d_keys(
        (result_type_equivalent*)(d_queryResultsTmp_),
        (result_type_equivalent*)(d_queryResults_));

    size_t tempStorageBytes = maxEncodeLength_*sizeof(encodedseq_t);
    void * d_tempStorage = (void*)(d_encodedSeq_);

    int numItems = numQueries_*maxResultsPerQuery_;
    // int numItems = h_resultOffsets_[numSegments_+1];
    // size_t tempStorageBytes = 255;
    hipError_t err = hipcub::DeviceSegmentedRadixSort::SortKeys(
        d_tempStorage, tempStorageBytes,
        d_keys,
        numItems, numSegments_,
        d_resultOffsets_, d_resultOffsets_ + 1,
        0, sizeof(result_type_equivalent) * CHAR_BIT,
        stream_
    );

    // hipStreamSynchronize(stream_);
    // CUERR

    if (err != hipSuccess) {                       \
        std::cout << "CUDA error: " << hipGetErrorString(err) << " : "    \
                  << __FILE__ << ", line " << __LINE__ << std::endl;       \
        exit(1);                                                           \
    }

    d_queryResults_    = (result_type*)d_keys.Current();
    d_queryResultsTmp_ = (result_type*)d_keys.Alternate();


    hipEventSynchronize(event_);

    hipMemcpyAsync(h_queryResults_, d_queryResults_,
        // numQueries_*maxResultsPerQuery_*sizeof(result_type),
        h_resultOffsets_[numSegments_]*sizeof(result_type),
        hipMemcpyDeviceToHost, stream_);


    //TODO max cand as template?
    #define MAX_CANDIDATES 8

    const size_t numBlocks = SDIV(numSegments_, 32);

    generate_top_candidates<MAX_CANDIDATES><<<numBlocks,32,0,stream_>>>(
        numSegments_,
        d_resultOffsets_,
        d_queryResults_,
        d_maxWindowsInRange_,
        d_topCandidates_);

    hipStreamSynchronize(stream_);
}


//---------------------------------------------------------------
template class query_batch<location>;

} // namespace mc
