
#include "query_batch.cuh"
#include "sketch_database.h"

namespace mc {


//---------------------------------------------------------------
template<class result_type>
query_batch<result_type>::query_batch(
    size_t maxQueries,
    size_t maxEncodeLength,
    size_t maxResultsPerQuery) :
    numQueries_{0},
    maxQueries_{maxQueries},
    maxEncodeLength_{maxEncodeLength},
    maxResultsPerQuery_{maxResultsPerQuery}
{
    if(maxQueries_) {
        hipHostMalloc(&queryIds_, maxQueries_*sizeof(query_id));

        hipHostMalloc(&h_encodeOffsets_, (maxQueries_+1)*sizeof(encodinglen_t));
        h_encodeOffsets_[0] = 0;
        hipHostMalloc(&h_queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));

        hipMalloc(&d_encodeOffsets_, (maxQueries_+1)*sizeof(encodinglen_t));
        hipMalloc(&d_queryResults_, maxQueries_*maxResultsPerQuery_*sizeof(result_type));
    }
    if(maxEncodeLength_) {
        hipHostMalloc(&h_encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));
        hipHostMalloc(&h_encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));

        hipMalloc(&d_encodedSeq_, maxEncodeLength_*sizeof(encodedseq_t));
        hipMalloc(&d_encodedAmbig_, maxEncodeLength_*sizeof(encodedambig_t));
    }
    CUERR

    hipStreamCreate(&stream_);
    CUERR
}
//---------------------------------------------------------------
template<class result_type>
query_batch<result_type>::~query_batch() {
    if(maxQueries_) {
        hipHostFree(queryIds_);

        hipHostFree(h_encodeOffsets_);
        hipHostFree(h_queryResults_);

        hipFree(d_encodeOffsets_);
        hipFree(d_queryResults_);
    }
    if(maxEncodeLength_) {
        hipHostFree(h_encodedSeq_);
        hipHostFree(h_encodedAmbig_);

        hipFree(d_encodedSeq_);
        hipFree(d_encodedAmbig_);
    }
    CUERR

    hipStreamDestroy(stream_);
    CUERR
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::copy_queries_to_device_async() {
    hipMemcpyAsync(d_encodeOffsets_, h_encodeOffsets_,
                    (numQueries_+1)*sizeof(encodinglen_t),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_encodedSeq_, h_encodedSeq_,
                    h_encodeOffsets_[numQueries_]*sizeof(encodedseq_t),
                    hipMemcpyHostToDevice, stream_);
    hipMemcpyAsync(d_encodedAmbig_, h_encodedAmbig_,
                    h_encodeOffsets_[numQueries_]*sizeof(encodedambig_t),
                    hipMemcpyHostToDevice, stream_);
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::copy_results_to_host_async() {
    hipMemcpyAsync(h_queryResults_, d_queryResults_,
                    numQueries_*maxResultsPerQuery_*sizeof(result_type),
                    hipMemcpyHostToDevice, stream_);
}


//---------------------------------------------------------------
template<class result_type>
void query_batch<result_type>::sync_stream() {
    hipStreamSynchronize(stream_);
}



//---------------------------------------------------------------
template class query_batch<location>;

} // namespace mc
